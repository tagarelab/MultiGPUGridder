#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"
#include <math.h>       /* round, floor, ceil, trunc */
 
__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter = volSize/2;
    int imgCenter = imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;   

    __shared__ float locKer[1000];

    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        {
            locKer[kerIndex]=*(ker+kerIndex);
            // locKer[kerIndex]=1;//test
        }
    }
    __syncthreads();      

    // Are we inside the image bounds?
    if ( i < 0 || i > volSize || j < 0 || j > volSize)
    {
        return;
    }

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;

            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        // w = 1;
                        // test
                        // *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+
                                //  ( (1+k1*volSize*volSize+j1*volSize+i1));


                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));

                        // }
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i
}

void gpuForwardProject::Execute()
{ 
    std::cout << "Running gpuForwardProject()..." << '\n';

    // For compactness define the CASImgSize, CASVolSize, and ImgSize here
    int CASImgSize = this->d_CASImgs->GetSize(0);
    int CASVolSize = this->d_CASVolume->GetSize(0);
    int ImgSize = this->d_Imgs->GetSize(0);    
    
    // Allocate temporary hipfftComplex arrays
    hipfftComplex *d_CASImgsComplex;
    hipMalloc(&d_CASImgsComplex, sizeof(hipfftComplex) * CASImgSize * CASImgSize * std::min(this->nAxes, this->MaxAxesAllocated));

    // Set the current GPU device to run the kernel
    hipSetDevice(this->GPU_Device);    

    // Create the CUDA streams  
    hipStream_t * streams = (hipStream_t *)malloc(sizeof(hipStream_t) * this->nStreams);
    
    for (int i = 0; i < this->nStreams; i++) // Loop through the streams
    {             
        hipStreamCreate(&streams[i]);
    }

    // Define CUDA kernel dimensions
    dim3 dimGrid(this->gridSize, this->gridSize, 1);
    dim3 dimBlock(this->blockSize, this->blockSize, 1);

    // Initilize a variable which will remember how many axes have been assigned to the GPU
	// during the current batch. This is needed for calculating the offset for the stream when 
	// the number of streams is greater than the number of GPUs. This resets to zeros after each
	// batch since the same GPU memory is used for the next batch.
	int numAxesGPU_Batch = 0;
    
	// How many coordinate axes to assign to each stream?
	int numAxesPerStream;
	if (this->nAxes <= this->MaxAxesAllocated)
	{
		// The number of coordinate axes is less than or equal to the total number of axes to process
		numAxesPerStream = ceil((double)this->nAxes / (double)this->nStreams);
	}
	else
	{
		// Several batches will be needed so evenly split the MaxAxesAllocated by the number of streams
		numAxesPerStream = ceil((double)this->MaxAxesAllocated / (double)this->nStreams);
    }	

    int processed_nAxes = 0; // Cumulative number of axes which have already been assigned to a CUDA stream

	// While we have coordinate axes to process, loop through the GPUs and the streams
	int MaxBatches = 1000; // Maximum iterations in case we get stuck in the while loop for some reason
	int batch = 0;

	while (processed_nAxes < this->nAxes && batch < MaxBatches)
	{
        Log2(" ", " ");
        Log2(" ", " ");
        Log2(" ", " ");
        Log2("Batch: ", batch);
        
		for (int i = 0; i < this->nStreams; i++) // Loop through the streams 
		{   
            numAxesGPU_Batch = 0; // test

            Log2(" ", " ");
            Log2("Stream: ", i);
            Log2("processed_nAxes", processed_nAxes);
            Log2("numAxesPerStream", numAxesPerStream);
            Log2("numAxesGPU_Batch", numAxesGPU_Batch);
                        
            
            // If we're about to process more than the number of coordinate axes, process the remaining faction of numAxesPerStream
			if (processed_nAxes + numAxesPerStream >= this->nAxes)
			{
				// Process the remaining fraction of numAxesPerStream
				numAxesPerStream = min(numAxesPerStream, nAxes - processed_nAxes);
			}

			// Check to make sure we don't try to process more coordinate axes than we have and that we have at least one axes to process
			if (processed_nAxes + numAxesPerStream > this->nAxes || numAxesPerStream < 1)
			{
				return;
            }

			// Calculate the offsets (in bytes) to determine which part of the array to copy for this stream
			int CoordAxes_CPU_Offset = processed_nAxes * 9;  // Each axes has 9 elements (X, Y, Z)
			int coord_Axes_CPU_streamBytes = numAxesPerStream * 9 * sizeof(float);

			// Use the number of axes already assigned to this GPU since starting the current batch to calculate the currect offset			
            int gpuCASImgs_Offset          = numAxesGPU_Batch * CASImgSize * CASImgSize;
            int gpuImgs_Offset             = numAxesGPU_Batch * ImgSize * ImgSize;
            int gpuCoordAxes_Stream_Offset = numAxesGPU_Batch * 9;

        	// Copy the section of gpuCoordAxes which this stream will process on the current GPU
            hipMemcpyAsync(
                this->d_CoordAxes->GetPointer(gpuCoordAxes_Stream_Offset), //&this->d_CoordAxes[gpuCoordAxes_Stream_Offset], 
                &coordAxes_CPU_Pinned[CoordAxes_CPU_Offset],
                coord_Axes_CPU_streamBytes,
                hipMemcpyHostToDevice, streams[i]);                
                
            // Run the forward projection kernel     
			gpuForwardProjectKernel <<< dimGrid, dimBlock, 0, streams[i] >> > (
                this->d_CASVolume->GetPointer(),
                CASVolSize,
                this->d_CASImgs->GetPointer(gpuCASImgs_Offset),
                CASImgSize,
                this->d_CoordAxes->GetPointer(gpuCoordAxes_Stream_Offset),
                numAxesPerStream, 
                this->maskRadius,
                this->d_KB_Table->GetPointer(),
                this->d_KB_Table->GetSize(0),
                this->kerHWidth);

            // Optionally: Copy the resulting CAS images back to the host pinned memory (CPU)
            if (this->CASImgs_CPU_Pinned != NULL)
            {
                // Have to use unsigned long long since the array may be longer than the max value int32 can represent
                // imgSize is the size of the zero padded projection images
                unsigned long long *CASImgs_CPU_Offset = new  unsigned long long[3];
                CASImgs_CPU_Offset[0] = (unsigned long long)( CASImgSize );
                CASImgs_CPU_Offset[1] = (unsigned long long)( CASImgSize );
                CASImgs_CPU_Offset[2] = (unsigned long long)( processed_nAxes );

                // How many bytes are the output images?
                int gpuCASImgs_streamBytes = CASImgSize * CASImgSize * numAxesPerStream * sizeof(float);

                hipMemcpyAsync(
                    &CASImgs_CPU_Pinned[CASImgs_CPU_Offset[0] * CASImgs_CPU_Offset[1] * CASImgs_CPU_Offset[2]],
                    this->d_CASImgs->GetPointer(gpuCASImgs_Offset),
                    gpuCASImgs_streamBytes,
                    hipMemcpyDeviceToHost,
                    streams[i]);               
            }

            // Convert the CAS projection images back to images using an inverse FFT and cropping out the zero padding
            gpuFFT::CASImgsToImgs(
                streams[i],
                CASImgSize,
                ImgSize,
                this->d_CASImgs->GetPointer(gpuCASImgs_Offset),
                this->d_Imgs->GetPointer(gpuImgs_Offset),                
                &d_CASImgsComplex[gpuCASImgs_Offset],
                numAxesPerStream);   

            // Have to use unsigned long long since the array may be longer than the max value int32 can represent
			// imgSize is the size of the zero padded projection images
			unsigned long long *Imgs_CPU_Offset = new  unsigned long long[3];
			Imgs_CPU_Offset[0] = (unsigned long long)(ImgSize);
			Imgs_CPU_Offset[1] = (unsigned long long)(ImgSize);
			Imgs_CPU_Offset[2] = (unsigned long long)(processed_nAxes);
            
			// How many bytes are the output images?
			int gpuImgs_streamBytes = ImgSize * ImgSize * numAxesPerStream * sizeof(float);
                
            // Lastly, copy the resulting cropped projection images back to the host pinned memory (CPU)            
            hipMemcpyAsync(
				Imgs_CPU_Pinned + Imgs_CPU_Offset[0] * Imgs_CPU_Offset[1] * Imgs_CPU_Offset[2],
                this->d_Imgs->GetPointer(gpuImgs_Offset),
                gpuImgs_streamBytes,
                hipMemcpyDeviceToHost,
                streams[i]);

			// Update the overall number of coordinate axes which have already been assigned to a CUDA stream
            processed_nAxes = processed_nAxes + numAxesPerStream;   

            // Update the number of axes which have been assigned to this GPU during the current batch
            numAxesGPU_Batch = numAxesGPU_Batch + numAxesPerStream;
            
        }
        
		// Increment the batch number
		batch++;

        // Reset the number of axes processed during the current batch variable
        numAxesGPU_Batch = 0;        

		// Synchronize before running the next batch
		// TO DO: Consider replacing with hipStreamWaitEvent or similar to prevent blocking of the CPU
        hipDeviceSynchronize(); // needed?           
    }
    
    // Free the temporary array
    hipFree(d_CASImgsComplex);

    return; 
}




