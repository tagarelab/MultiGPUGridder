#include "hip/hip_runtime.h"
#include "gpuFFT.h"

#define Log(x)                  \
    {                           \
        std::cout << x << '\n'; \
    }


// Constructor
gpuFFT::gpuFFT(/* args */)
{
}

// Deconstructor
gpuFFT::~gpuFFT()
{
}


__global__ void cufftShift_3D_slice_kernel(hipfftComplex* input, hipfftComplex* output, int N, int nSlices)
{
	// 3D Volume, 2D Slice, 1D Line
	int sLine = N;
	int sSlice = N * N;
	int sVolume = N * N * N;

	// Transformations Equations
	int sEq1 = (sVolume + sSlice + sLine) / 2;
	int sEq2 = (sVolume + sSlice - sLine) / 2;
	int sEq3 = (sVolume - sSlice + sLine) / 2;
	int sEq4 = (sVolume - sSlice - sLine) / 2;

	// Thread Index 2D
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Each thread will do all the slices for some X, Y position in the 3D matrix
	for (int zIndex = 0; zIndex < nSlices; zIndex++)
	{
		// Thread Index Converted into 1D Index
		int index = (zIndex * sSlice) + (yIndex * sLine) + xIndex;

		if (zIndex < N / 2)
		{
			if (xIndex < N / 2)
			{
				if (yIndex < N / 2)
				{
					// First Quad
					output[index].x = input[index + sEq1].x;
					output[index].y = input[index + sEq1].y;
				}
				else
				{
					// Third Quad
					output[index].x = input[index + sEq3].x;
					output[index].y = input[index + sEq3].y;
				}
			}
			else
			{
				if (yIndex < N / 2)
				{
					// Second Quad
					output[index].x = input[index + sEq2].x;
					output[index].y = input[index + sEq2].y;
				}
				else
				{
					// Fourth Quad
					output[index].x = input[index + sEq4].x;
					output[index].y = input[index + sEq4].y;
				}
			}
		}
		else
		{
			if (xIndex < N / 2)
			{
				if (yIndex < N / 2)
				{
					// First Quad
					output[index].x = input[index - sEq4].x;
					output[index].y = input[index - sEq4].y;
				}
				else
				{
					// Third Quad
					output[index].x = input[index - sEq2].x;
					output[index].y = input[index - sEq2].y;
				}
			}
			else
			{
				if (yIndex < N / 2)
				{
					// Second Quad
					output[index].x = input[index - sEq3].x;
					output[index].y = input[index - sEq3].y;
				}
				else
				{
					// Fourth Quad
					output[index].x = input[index - sEq1].x;
					output[index].y = input[index - sEq1].y;
				}
			}
		}
	}
}


template <typename T>
__global__ void cufftShift_3D_slice_kernel(T* data, int N, int nSlices)
{
    // In place FFT shift using GPU
    // Modified from https://raw.githubusercontent.com/marwan-abdellah/cufftShift/master/Src/CUDA/Kernels/in-place/cufftShift_3D_IP.cu
    // GNU Lesser General Public License

    // 3D Volume & 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;
    int sVolume = N * N * N;

    // Transformations Equations
    int sEq1 = (sVolume + sSlice + sLine) / 2;
    int sEq2 = (sVolume + sSlice - sLine) / 2;
    int sEq3 = (sVolume - sSlice + sLine) / 2;
    int sEq4 = (sVolume - sSlice - sLine) / 2;

    // Thread
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth  = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index 2D
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Are we within the image bounds?
    if (xIndex < 0 || xIndex >= N || yIndex < 0 || yIndex >= N )
    {
        return;
    }

    T regTemp;

	// Each thread will do all the slices for some X, Y position in the 3D matrix
	for (int zIndex = 0; zIndex < nSlices; zIndex++)
	{

        // Thread Index Converted into 1D Index
        int index = (zIndex * sSlice) + (yIndex * sLine) + xIndex;

        if (zIndex < N / 2)
        {
            if (xIndex < N / 2)
            {
                if (yIndex < N / 2)
                {
                    regTemp = data[index];

                    // First Quad
                    data[index] = data[index + sEq1];

                    // Fourth Quad
                    data[index + sEq1] = regTemp;
                }
                else
                {
                    regTemp = data[index];

                    // Third Quad
                    data[index] = data[index + sEq3];

                    // Second Quad
                    data[index + sEq3] = regTemp;
                }
            }
            else
            {
                if (yIndex < N / 2)
                {
                    regTemp = data[index];

                    // Second Quad
                    data[index] = data[index + sEq2];

                    // Third Quad
                    data[index + sEq2] = regTemp;
                }
                else
                {
                    regTemp = data[index];

                    // Fourth Quad
                    data[index] = data[index + sEq4];

                    // First Quad
                    data[index + sEq4] = regTemp;
                }
            }
        }
    }
}

__global__ void ComplexImgsToCASImgs(float* CASimgs, hipfftComplex* imgs, int imgSize)
{
	// CUDA kernel for converting CASImgs to imgs
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
	int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

	// Are we outside the bounds of the image?
	if (i >= imgSize || i < 0 || j >= imgSize || j < 0) {
		return;
	}

	// Each thread will do all the slices for position X and Y
	for (int k = 0; k < imgSize; k++)
	{
		// CASimgs is the same dimensions as imgs
		int ndx = i + j * imgSize + k * imgSize * imgSize;

		// Summation of the real and imaginary components
		CASimgs[ndx] = imgs[ndx].x + imgs[ndx].y;
	}

	return;
}

__global__ void CASImgsToComplexImgs(float* CASimgs, hipfftComplex* imgs, int imgSize, int nSlices)
{
    // CUDA kernel for converting CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the image?
    if (i >= imgSize || i < 0 || j >= imgSize || j < 0){
        return;
    }
    
    // Each thread will do all the slices for position X and Y
    for (int k = 0; k < nSlices; k++)  {        

        int ndx_1 = i + j * imgSize + k * imgSize * imgSize;
        
        // Skip the first row and first column
        if (i == 0 || j == 0)
        {
            // Real component
            imgs[ndx_1].x = 0;

            // Imaginary component
            imgs[ndx_1].y = 0;

            
        } else 
        {
            // Offset to skip the first row then subtract from the end of the matrix and add the offset where the particular image starts in CASimgs
            int ndx_2 = imgSize + imgSize * imgSize - (i + j * imgSize) + k * imgSize * imgSize;

            // Real component
            imgs[ndx_1].x = 0.5*(CASimgs[ndx_1] + CASimgs[ndx_2]);

            // Imaginary component
            imgs[ndx_1].y = 0.5*(CASimgs[ndx_1] - CASimgs[ndx_2]);

        }
    }

    return;
}

__global__ void ComplexToReal(hipfftComplex* ComplexImg, float* RealImg, int imgSize, int nSlices)
{
    // CUDA kernel for extracting the real component of a hipfftComplex and then save the real component to a float array

    // CUDA kernel for converting the CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the image?
    if (i >= imgSize || i < 0 || j >= imgSize || j < 0){
        return;
    }
    
    // Each thread will do all the slices for some position X and Y
    for (int k=0; k < nSlices; k++)
    {         
        // Get the linear index of the current position
        int ndx = i + j * imgSize + k * imgSize * imgSize;       

        RealImg[ndx] = ComplexImg[ndx].x;
    }
}

__global__ void PadVolumeKernel(float* input, float* output, int inputImgSize, int outputImgSize, int padding)
{
    // Zero pad a volume using the GPU

    // Index of the output (smaller) image
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row
    
    // Are we outside the bounds of the image?
    if (i >= inputImgSize || i < 0 || j >= inputImgSize || j < 0){
        return;
    }

    // // Iterate over the input image (i.e. the smaller image)
    for (int k = 0; k < inputImgSize; k++){  

        // Get the linear index of the output (smaller) image
        int ndx_1 = i + j * inputImgSize + k * inputImgSize * inputImgSize;   

        // Get the linear index of the output (larger) image    
        int ndx_2 = 
        (i + padding) + 
        (j + padding) * outputImgSize +
        (k + padding) * outputImgSize *  outputImgSize;  

        output[ndx_2] = input[ndx_1];
    }
}

void gpuFFT::PadVolume(float *inputVol, float * outputVol, int inputImgSize, int outputImgSize)
{
    // Pad a volume (of dimensions 3) with zeros
    // Note: Output volume is larger than the input volume

    // Check the input parameters
    if(inputImgSize <=0)
    {
        std::cerr << "CropVolume(): Invalid image size." << '\n';
    }

    // Create the output volume
    //float *outputVol = new float[outputImgSize * outputImgSize * outputImgSize];
    memset(outputVol, 0, outputImgSize * outputImgSize * outputImgSize * sizeof(float));

    // for (int i = 0; i < outputImgSize * outputImgSize * outputImgSize; i++)
    // {
    //     outputVol[i] = 0;
    // }

    // How much to add to each side?
    int padding = (outputImgSize - inputImgSize) / 2;

    std::cout << "PadVolume() padding: " << padding << '\n';

    // For very small matrix sizes it might be faster to use the CPU instead of the GPU
    bool use_gpu = true;

    if (use_gpu == true)
    {
        // Allocate GPU memory to hold the input and output arrays
        float *d_input; 
        hipMalloc(&d_input, sizeof(float) * inputImgSize * inputImgSize * inputImgSize);
        float *d_output; 
        hipMalloc(&d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize);

        // Copy the input volume to the device
        hipMemcpy(d_input, inputVol, sizeof(float) * inputImgSize * inputImgSize * inputImgSize, hipMemcpyHostToDevice);

        // Run kernel to pad the intput array
        int gridSize = 32;
        int blockSize = ceil(inputImgSize / gridSize);

        dim3 dimGridCrop(gridSize, gridSize, 1);
        dim3 dimBlockCrop(blockSize, blockSize, 1);

        PadVolumeKernel<<< dimGridCrop, dimBlockCrop >>>(d_input, d_output, inputImgSize, outputImgSize, padding);

        // Copy the result back to the host
        hipMemcpy(outputVol, d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize, hipMemcpyDeviceToHost);

        // Free the GPU memory
        hipFree(d_input);
        hipFree(d_output);

    } else 
    {
        // Iterate over the input image (i.e. the smaller image)
        for (int i = 0; i < inputImgSize; i++)
        {
            for (int j = 0; j < inputImgSize; j++)
            {
                for (int k = 0; k < inputImgSize; k++)
                {

                    int input_ndx = i + j*inputImgSize + k*inputImgSize*inputImgSize;

                    int output_ndx = (i + padding) + (j+padding)*outputImgSize + (k+padding)*outputImgSize*outputImgSize;

                    outputVol[output_ndx] = inputVol[input_ndx];
                }
            }
        }
    }
    
    return;

}

__global__ void CropImgs(float* input, float* output, int inputImgSize, int outputImgSize, int nSlices)
{
    // Given the final projection images, crop out the zero padding to reduce memory size and transfer speed back to the CPU
    // inputImgSize is the size of the CASImgs (i.e. larger)
    // outputImgSize is the size of the images (i.e. smaller)

    // Index of the output (smaller) image
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the smaller output image?
    if (i >= outputImgSize || i < 0 || j >= outputImgSize || j < 0){
        return;
    }

    // How much zero padding to remove from each side?
    int padding = (inputImgSize - outputImgSize) / 2;
    
    if (padding <= 0)
    {
        return;
    }

    for (int k = 0; k < nSlices; k++){

        // Get the linear index of the output (smaller) image
        int ndx_1 = i + j * outputImgSize + k * outputImgSize * outputImgSize;   

        // Get the linear index of the input (larger) image
        // NOTE: No padding in the Z direction because we are cropping each 2D images individually
        int ndx_2 = 
        (i + padding) + 
        (j + padding) * inputImgSize +
        k * inputImgSize *  inputImgSize;  

        output[ndx_1] = input[ndx_2];

    }
}

__global__ void NormalizeImgs(float* input, int ImgSize, int numImgs, int NormalizeFactor)
{
    // Normalize images by dividing each voxel by some normalization factor

    // Index of the image
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the smaller output image?
    if (i >= ImgSize || i < 0 || j >= ImgSize || j < 0){
        return;
    }

    for (int k = 0; k < numImgs; k++){

        // Get the linear index of image
        int ndx = i + j * ImgSize + k * ImgSize * ImgSize;   

        input[ndx] = input[ndx] / NormalizeFactor;

    }
}


void gpuFFT::VolumeToCAS(float* inputVol, int inputVolSize, float* outputVol, int interpFactor, int extraPadding)
{
    // Convert a CUDA array to CAS array
    // Note: The volume must be square (i.e. have the same dimensions for the X, Y, and Z)
    // Step 1: Pad with zeros
    // Step 2: fftshift
    // Step 3: Take discrete Fourier transform using cuFFT
    // Step 4: fftshift
    // Step 5: Convert to CAS volume using CUDA kernel
    
    std::cout << "gpuFFT::VolumeToCAS()" << '\n';

    // STEP 1
    // Example: input size = 128; interpFactor = 2; paddedVolSize = 256
    int paddedVolSize = inputVolSize * interpFactor;

    Log("VolumeToCAS():");
    Log(inputVolSize);
    Log(interpFactor);
    Log(extraPadding);
    Log(paddedVolSize);

    // Pad the input volume with zeros
    float* inputVol_Padded = new float[paddedVolSize * paddedVolSize * paddedVolSize];
    PadVolume(inputVol, inputVol_Padded, inputVolSize, paddedVolSize);
    
    // Plan the forward FFT
    hipfftHandle forwardFFTPlan;           
    hipfftPlan3d(&forwardFFTPlan, paddedVolSize, paddedVolSize, paddedVolSize, HIPFFT_C2C);

    int array_size = paddedVolSize * paddedVolSize * paddedVolSize;
    
    // Allocate memory for the resulting CAS volumes
    float * d_CAS_Vol, *h_CAS_Vol;
    hipMalloc(&d_CAS_Vol, sizeof(float) * array_size);
    h_CAS_Vol = (float *) malloc(sizeof(float) * array_size);

    // Create temporary arrays to hold the hipfftComplex array        
    hipfftComplex *h_complex_array, *d_complex_array;//, *d_complex_output_array;
    hipMalloc(&d_complex_array, sizeof(hipfftComplex) * array_size);
    // hipMalloc(&d_complex_output_array, sizeof(hipfftComplex) * array_size);
    h_complex_array = (hipfftComplex *) malloc(sizeof(hipfftComplex) * array_size);
    
    // Convert the padded volume to a hipfftComplex array
    // TO DO: Replace this with the CUDA kernel
    for (int k = 0; k < array_size; k++) {
        h_complex_array[k].x = inputVol_Padded[k]; // Real component
        h_complex_array[k].y = 0;                  // Imaginary component
    }

    // Copy the complex version of the GPU volume to the first GPU
    hipMemcpy( d_complex_array, h_complex_array, array_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);        

    int gridSize  = ceil(paddedVolSize / 32);
    int blockSize = 32; // i.e. 32*32 threads

    // Define CUDA kernel dimensions for converting the complex volume to a CAS volume
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    // STEP 2
    // Apply an in place 3D FFT Shift
    cufftShift_3D_slice_kernel<<< dimGrid, dimBlock >>> (d_complex_array, paddedVolSize, paddedVolSize);
    // cufftShift_3D_slice_kernel <<< dimGrid, dimBlock >>> (d_complex_array, d_complex_output_array, paddedVolSize, paddedVolSize);

    // STEP 3
    // Execute the forward FFT on the 3D array
    hipfftExecC2C(forwardFFTPlan, (hipfftComplex *) d_complex_array, (hipfftComplex *) d_complex_array, HIPFFT_FORWARD);

    // STEP 4
    // Apply a second in place 3D FFT Shift
    cufftShift_3D_slice_kernel<<< dimGrid, dimBlock>>> (d_complex_array, paddedVolSize, paddedVolSize);
    // cufftShift_3D_slice_kernel <<< dimGrid, dimBlock >>> (d_complex_output_array, d_complex_array, paddedVolSize, paddedVolSize);

    // STEP 5
    // Convert the complex result of the forward FFT to a CAS img type
    ComplexImgsToCASImgs<<< dimGrid, dimBlock >>>(
        d_CAS_Vol, d_complex_array, paddedVolSize
    );
    
    // Copy the resulting CAS volume back to the host
    hipMemcpy(h_CAS_Vol, d_CAS_Vol, array_size * sizeof(float), hipMemcpyDeviceToHost);        

    // Wait for the stream to finish copying the result back to the host
    // hipStreamSynchronize(Stream);
    hipDeviceSynchronize(); // TO DO: replace with stream sync

    // STEP 6
    // Pad the result with the additional padding
    // Example: input size = 128; interpFactor = 2; extra padding = 3; -> paddedVolSize_Extra= 262
    int paddedVolSize_Extra = paddedVolSize + extraPadding * 2;

    // Pad the padded volume with the extra zero padding
    // TO DO: make PadVolume completely on the GPU to remove the stream sync step above
    PadVolume(h_CAS_Vol, outputVol, paddedVolSize, paddedVolSize_Extra);

    // Free the temporary memory
    hipFree(d_complex_array);
    hipFree(d_CAS_Vol);   
    std::free(inputVol_Padded);
    std::free(h_CAS_Vol);

    std::cout << "gpuFFT::VolumeToCAS() Done" << '\n';

    // Return the resulting CAS volume
    return;

}

void gpuFFT::CASImgsToImgs(
    hipStream_t& stream, int gridSize, int blockSize, int CASImgSize, 
    int ImgSize, float* d_CASImgs, float* d_imgs, int numImgs)
{

    hipDeviceSynchronize();

    // test
    // numImgs = 400;    

    hipSetDevice(0);   

    // Convert a CAS images array to images
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    std::cout << "CASImgsToImgs()" << '\n';
    std::cout << "gridSize: " << gridSize << '\n';
    std::cout << "blockSize: " << blockSize << '\n';
    std::cout << "CASImgSize: " << CASImgSize << '\n';
    std::cout << "ImgSize: " << ImgSize << '\n';
    std::cout << "numImgs: " << numImgs << '\n';
    std::cout << "sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs: " << sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs << '\n';



    // Allocate a temporary hipfftComplex array 
    hipfftComplex *d_CASImgsComplex2;
    hipMalloc(&d_CASImgsComplex2, sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs);

    hipfftComplex *d_CASImgsComplex2Output;
    hipMalloc(&d_CASImgsComplex2Output, sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs);


    CASImgsToComplexImgs<<< dimGrid, dimBlock >>>(d_CASImgs, d_CASImgsComplex2, CASImgSize, numImgs);

    hipMemset(d_CASImgs, 0, sizeof(float)*CASImgSize*CASImgSize*numImgs); 

    cufftShift_3D_slice_kernel <<< dimGrid, dimBlock >>> (d_CASImgsComplex2, d_CASImgsComplex2Output, CASImgSize, numImgs);
 

    hipDeviceSynchronize();



    // // Execute the forward FFT on each 2D array
    // for (int i = 0; i<numImgs; i++)
    // {
    //     // std::cout << "Inverse FFT " << i << '\n';
    //     // std::cout << "i*CASImgSize*CASImgSize: " << i*CASImgSize*CASImgSize << '\n';

    //     // std::cout << "Left: " << CASImgSize * CASImgSize * numImgs  - i*CASImgSize*CASImgSize << '\n';

    //     // Plan the inverse FFT
    //     hipfftHandle inverseFFTPlan;           
    //     hipfftPlan2d(&inverseFFTPlan, CASImgSize, CASImgSize, HIPFFT_C2C);

    //     hipfftExecC2C(inverseFFTPlan,
    //         &*(d_CASImgsComplex2Output + i*CASImgSize*CASImgSize), 
    //         &*(d_CASImgsComplex2 + i*CASImgSize*CASImgSize),
    //         HIPFFT_BACKWARD);
            
    //     // std::cout << "result: " << result << '\n';
    //     hipDeviceSynchronize();

    //     hipfftDestroy(inverseFFTPlan);
    // }


        // Create a plan for taking the inverse of the CAS imgs
        hipfftHandle inverseFFTPlan;   
        int nRows = CASImgSize;
        int nCols = CASImgSize;
        int batch = numImgs;            // --- Number of batched executions
        int rank = 2;                   // --- 2D FFTs
        int n[2] = {nRows, nCols};      // --- Size of the Fourier transform
        int idist = nRows*nCols;        // --- Distance between batches
        int odist = nRows*nCols;        // --- Distance between batches
    
        int inembed[] = {nRows, nCols}; // --- Input size with pitch
        int onembed[] = {nRows, nCols}; // --- Output size with pitch
    
        int istride = 1;                // --- Distance between two successive input/output elements
        int ostride = 1;                // --- Distance between two successive input/output elements
        
        hipDeviceSynchronize();
        hipfftPlanMany(&inverseFFTPlan,  rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);            
        // hipfftSetStream(inverseFFTPlan, stream); // Set the FFT plan to the current stream to process
        // hipDeviceSynchronize();
    
        // Inverse FFT
        hipfftExecC2C(inverseFFTPlan, (hipfftComplex *) d_CASImgsComplex2Output, (hipfftComplex *) d_CASImgsComplex2, HIPFFT_BACKWARD);
    
   
        



   

    cufftShift_3D_slice_kernel <<< dimGrid, dimBlock >>> (d_CASImgsComplex2, d_CASImgsComplex2Output, CASImgSize, numImgs);

    ComplexToReal<<< dimGrid, dimBlock >>>(d_CASImgsComplex2Output, d_CASImgs, CASImgSize, numImgs);    

    CropImgs<<< dimGrid, dimBlock >>>(d_CASImgs, d_imgs, CASImgSize, ImgSize, numImgs);
    NormalizeImgs<<< dimGrid, dimBlock >>>(d_imgs, ImgSize, numImgs, CASImgSize * CASImgSize);
    
    return;










    // Check to make sure the GPU has enough available memory left
    size_t mem_tot_0 = 0;
    size_t mem_free_0 = 0;
    hipMemGetInfo(&mem_free_0, &mem_tot_0);
    std::cout << "Free GPU memory: " << mem_free_0 << " out of " << mem_tot_0 << '\n';


    // hipMemset(d_CASImgsComplex, 0, sizeof(hipfftComplex)*CASImgSize*CASImgSize*numImgs) ;
    // hipMemset(d_imgs, 0, sizeof(float)*ImgSize*ImgSize*numImgs) ;

    // Allocate a temporary hipfftComplex array 
    hipfftComplex *d_CASImgsComplex, *d_CASImgsComplexOutput;
    hipMalloc(&d_CASImgsComplex, sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs);
    hipMalloc(&d_CASImgsComplexOutput, sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs);

    hipMemset(d_CASImgsComplexOutput, 0, sizeof(hipfftComplex)*CASImgSize*CASImgSize*numImgs) ;
    hipDeviceSynchronize();


    // Convert the CASImgs to complex cufft type
    CASImgsToComplexImgs<<< dimGrid, dimBlock, 0, stream >>>(d_CASImgs, d_CASImgsComplex, CASImgSize, numImgs);

    hipDeviceSynchronize();

    // Run FFTShift on d_CASImgsComplex
    cufftShift_3D_slice_kernel <<< dimGrid, dimBlock, 0, stream >>> (d_CASImgsComplex, CASImgSize, numImgs);

    hipDeviceSynchronize();

    // // Create a plan for taking the inverse of the CAS imgs
    // hipfftHandle inverseFFTPlan;   
    // int nRows = CASImgSize;
    // int nCols = CASImgSize;
    // int batch = numImgs;            // --- Number of batched executions
    // int rank = 2;                   // --- 2D FFTs
    // int n[2] = {nRows, nCols};      // --- Size of the Fourier transform
    // int idist = nRows*nCols;        // --- Distance between batches
    // int odist = nRows*nCols;        // --- Distance between batches

    // int inembed[] = {nRows, nCols}; // --- Input size with pitch
    // int onembed[] = {nRows, nCols}; // --- Output size with pitch

    // int istride = 1;                // --- Distance between two successive input/output elements
    // int ostride = 1;                // --- Distance between two successive input/output elements
    
    // hipDeviceSynchronize();
    // // hipfftPlanMany(&inverseFFTPlan,  rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);            
    // // hipfftSetStream(inverseFFTPlan, stream); // Set the FFT plan to the current stream to process
    // // hipDeviceSynchronize();

    // // Inverse FFT
    // // hipfftExecC2C(inverseFFTPlan, (hipfftComplex *) d_CASImgsComplex, (hipfftComplex *) d_CASImgsComplex, HIPFFT_BACKWARD);






    // TEST
    // hipfftComplex * h_CASImgsComplex = new hipfftComplex[CASImgSize*CASImgSize*numImgs];

    // hipMemcpy(h_CASImgsComplex, d_CASImgsComplex, CASImgSize*CASImgSize*numImgs*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // // Make the entire image equal to a scalar
    // for (int i=0; i<CASImgSize*CASImgSize*numImgs; i++)
    // {
    //     h_CASImgsComplex[i].x = 2;   
    // }

    // hipMemcpy(d_CASImgsComplex, h_CASImgsComplex, CASImgSize*CASImgSize*numImgs*sizeof(hipfftComplex), hipMemcpyHostToDevice);
    // END TEST


    // STEP 3
    // Execute the forward FFT on each 2D array
    for (int i = 0; i<numImgs; i++)
    {
        std::cout << "Inverse FFT " << i << '\n';

        // Check to make sure the GPU has enough available memory left
        size_t mem_tot_0 = 0;
        size_t mem_free_0 = 0;
        hipMemGetInfo(&mem_free_0, &mem_tot_0);
        std::cout << "Free GPU memory: " << mem_free_0 << " out of " << mem_tot_0 << '\n';



        // Plan the inverse FFT
        hipfftHandle inverseFFTPlan;           
        hipfftPlan2d(&inverseFFTPlan, CASImgSize, CASImgSize, HIPFFT_C2C);
        hipfftSetStream(inverseFFTPlan, stream); // Set the FFT plan to the current stream to process

        hipfftResult_t result = hipfftExecC2C(inverseFFTPlan,
            (hipfftComplex *) &d_CASImgsComplex[i*CASImgSize*CASImgSize], // &d_CASImgsComplex[i*CASImgSize*CASImgSize]
            (hipfftComplex *) &d_CASImgsComplexOutput[i*CASImgSize*CASImgSize],
            HIPFFT_BACKWARD);
        
        std::cout << "result: " << result << '\n';
        hipDeviceSynchronize();

        hipfftDestroy(inverseFFTPlan);
    }

    // FFTShift again on d_CASImgsComplex
    cufftShift_3D_slice_kernel <<< dimGrid, dimBlock, 0, stream>>> (d_CASImgsComplexOutput, CASImgSize, numImgs);

    // test
    // hipMemset(d_CASImgsComplex, 0, sizeof(hipfftComplex) * CASImgSize * CASImgSize * numImgs);
    // hipMemset(d_CASImgs, 0, sizeof(float)*CASImgSize*CASImgSize*numImgs) ; // TEST TEST

    // Convert from the complex images to the real (resue the d_CASImgs GPU array)
    ComplexToReal<<< dimGrid, dimBlock, 0, stream >>>(d_CASImgsComplexOutput, d_CASImgs, CASImgSize, numImgs);            

    hipDeviceSynchronize();

    // Run kernel to crop the projection images (to remove the zero padding)   
    std::cout << "CASImgSize CASImgSize CASImgSize: " << CASImgSize << '\n';
    std::cout << "ImgSize ImgSize ImgSize: " << ImgSize << '\n';

    CropImgs<<< dimGrid, dimBlock, 0, stream >>>(d_CASImgs, d_imgs, CASImgSize, ImgSize, numImgs);

    // Run kernel to normalize the projection images (by dividing by the CASImgsize times CASImgSize)  
    // This scaling is introduced during the FFT 
    NormalizeImgs<<< dimGrid, dimBlock, 0, stream >>>(d_imgs, ImgSize, numImgs, CASImgSize * CASImgSize);
}