#include "hip/hip_runtime.h"
#include "DivideScalarFilter.h"

__global__ void DivideScalarKernel(float *Input, float Scalar, int VolumeSize, int nSlices)
{
    // CUDA kernel for converting a real image to a complex type

    // CUDA kernel for converting the CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the image?
    if (i >= VolumeSize || i < 0 || j >= VolumeSize || j < 0)
    {
        return;
    }

    // Each thread will do all the slices for some position X and Y
    for (int k = 0; k < nSlices; k++)
    {
        // Get the linear index of the current position
        int ndx = i + j * VolumeSize + k * VolumeSize * VolumeSize;

        Input[ndx] = Input[ndx] / Scalar;
    }
}

void DivideScalarFilter::UpdateFilter(float *Input, hipStream_t *stream)
{
    // Divide a CUDA array by some integer scalar

    // Check the input parameters
    if (this->VolumeSize <= 0)
    {
        std::cerr << "Error DivideScalarFilter(): Volume size parameter was not set. Please use SetVolumeSize() function first." << '\n';
        return;
    } else if (this->Scalar == 0)
    {
        std::cerr << "Error DivideScalarFilter(): Scalar parameter was not set. Please use SetScalar() function first." << '\n';
        return;
    }

    // Running a volume if the number of slices is not specified
    int NumberSlices;
    if (this->nSlices <= 0)
    {
        NumberSlices = this->VolumeSize;
    }
    else
    {
        // Running a stack of 2D images
        NumberSlices = this->nSlices;
    }

    // Define CUDA kernel launch dimensions
    // Iterate over the X,Y positions for all slices
    int Grid = ceil(double(this->VolumeSize) / double(32));
    Grid = max(Grid, 1); // At least one

    dim3 dimGrid(Grid, Grid, 1);
    dim3 dimBlock(32, 32, 1); // i.e. 32*32 threads

    // Use the CUDA stream if one was provided
    if (stream != NULL)
    {
        DivideScalarKernel<<<dimGrid, dimBlock, 0, *stream>>>(Input, this->Scalar, this->VolumeSize, NumberSlices);
    }
    else
    {
        DivideScalarKernel<<<dimGrid, dimBlock>>>(Input, this->Scalar, this->VolumeSize, NumberSlices);
    }

    return;
};
