#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{

    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter= volSize/2;
    int imgCenter=imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;
   

    __shared__ float locKer[1000];

    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        locKer[kerIndex]=*(ker+kerIndex);
    }
    __syncthreads();   
   

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;


            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        //w=expf(-(ri*ri+rj*rj+rk*rk)/(2*sigma*sigma));
                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+//w;
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i

}


void gpuForwardProject(
    std::vector<float*> gpuVol_Vector, std::vector<float*> gpuCASImgs_Vector,       // Vector of GPU array pointers
    std::vector<float*> gpuCoordAxes_Vector, std::vector<float*> ker_bessel_Vector, // Vector of GPU array pointers
    float * CASImgs_CPU_Pinned, float * coordAxes_CPU_Pinned, // Pointers to pinned CPU arrays for input / output
    int volSize, int imgSize, int nAxes, float maskRadius, int kerSize, float kerHWidth // Parameters and constants
)
{
  
    int nStreams = 1; // One stream for each GPU for now
    int numGPUs  = 4;  // TO DO: make this an input variable

    // Check the input vector sizes first
    if (gpuVol_Vector.size() != nStreams || gpuCASImgs_Vector.size() != nStreams || gpuCoordAxes_Vector.size() != nStreams || ker_bessel_Vector.size() != nStreams)
    {
        // std::cerr << "gpuForwardProject(): Input GPU pointer sizes is not equal to the number of CUDA streams." << '\n';
        // return;
    }

    std::cout << "gpuVol_Vector.size(): " << gpuVol_Vector.size() << '\n';
    std::cout << "gpuCASImgs_Vector.size(): " << gpuCASImgs_Vector.size() << '\n';
    std::cout << "gpuCoordAxes_Vector.size(): " << gpuCoordAxes_Vector.size() << '\n';
    std::cout << "ker_bessel_Vector.size(): " << ker_bessel_Vector.size() << '\n';

    // Try without the streams first 

    int i = 0;


    // How many bytes is each array
    int coord_Axes_streamBytes = 2034 * sizeof(float);//nAxes * 9 * sizeof(float); // Copy the entire vector for now
    int gpuCASImgs_streamBytes = 128* 128 * 226 * sizeof(float); // Copy the entire array for now

    std::cout << "coord_Axes_streamBytes: " << coord_Axes_streamBytes << '\n';
    std::cout << "gpuCASImgs_streamBytes: " << gpuCASImgs_streamBytes << '\n';

    std::cout << "coordAxes_CPU_Pinned: " << coordAxes_CPU_Pinned << '\n';
    std::cout << "gpuVol_Vector[i]: " << gpuVol_Vector[i] << '\n';
    std::cout << "gpuCASImgs_Vector[i]: " << gpuCASImgs_Vector[i] << '\n';
    std::cout << "gpuCoordAxes_Vector[i]: " << gpuCoordAxes_Vector[i] << '\n';
    std::cout << "ker_bessel_Vector[i]: " << ker_bessel_Vector[i] << '\n';

  

    hipMemcpy(gpuCoordAxes_Vector[i], coordAxes_CPU_Pinned, coord_Axes_streamBytes, hipMemcpyHostToDevice);
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk( hipDeviceSynchronize() );

    // Run the forward projection kernel
    dim3 dimGrid(32, 32, 1);
    dim3 dimBlock(4, 4, 1);

    hipSetDevice(0);
    gpuForwardProjectKernel<<< dimGrid, dimBlock >>>(
        gpuVol_Vector[i], 134, gpuCASImgs_Vector[i],
         128, gpuCoordAxes_Vector[i], nAxes,
         63, ker_bessel_Vector[i], 501, 2);

    
    gpuErrchk( hipDeviceSynchronize() );

    // gpuForwardProjectKernel<<< dimGrid, dimBlock >>>(vol, volSize, img, imgSize, axes, nAxes, maskRadius,ker, kerSize, kerHWidth);

    gpuErrchk( hipPeekAtLastError() );

    // Copy the resulting gpuCASImgs to the host (CPU)
    // TO DO: Only copy a subset of this
    hipMemcpy(CASImgs_CPU_Pinned, gpuCASImgs_Vector[i], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost);
    gpuErrchk( hipPeekAtLastError() );


    gpuErrchk( hipDeviceSynchronize() );


    std::cout << "Done with gpuForwardProjectKernel" << '\n';

    return; 









   	// // Create some CUDA streams
    // hipStream_t stream[nStreams];
    // for (int i = 0; i < nStreams; i++){

    //     // Split streams by GPU
    //     int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs

    //     if (curr_GPU <= numGPUs)
    //     {
    //         hipSetDevice(curr_GPU);
    //         hipStreamCreate(&stream[i]);
    //     } else 
    //     {
    //         std::cerr << "gpuForwardProject(): Failed to create CUDA stream." << '\n';
    //         return;
    //     }        
    // }       

    // // How many bytes is each async streaming?
    // int coord_Axes_streamBytes = nAxes * 9 * sizeof(float); // Copy the entire vector for now
    // int gpuCASImgs_streamBytes = 128 * 128 * nAxes * 9 * sizeof(float); // Copy the entire array for now

    // std::cout << "coord_Axes_streamBytes: " << coord_Axes_streamBytes << '\n';
    // std::cout << "gpuCASImgs_streamBytes: " << gpuCASImgs_streamBytes << '\n';


    // gpuErrchk( hipDeviceSynchronize() ); // Probably not needed

    // // Setup the CUDA streams now
    // for (int i = 0; i < nStreams; ++i){
        
    //     // TO DO: Is it necessary to use hipSetDevice() here?
    //     int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs

    //     curr_GPU = 0; // For now
    //     hipSetDevice(curr_GPU);
        
    //     // Get the GPU pointers for this strea
    //     // float *devPtr_gpuVol     = gpuVol_V    // std::cout << "Done with gpuForwardProjectKernel" << '\n';ctor[i];
    //     // float *devPtr_gpuCASImgs = gpuCASImgs_Vector[i];
    //     // float *devPtr_Coord_Axes = gpuCoordAxes_Vector[i];
    //     // float *devPtr_ker_bessel = ker_bessel_Vector[i];

    //     // TO DO: Only copy a subset of the array

    //     // Copy coord axes from pinned host (CPU) to device (GPU)
    //     hipMemcpyAsync(&gpuCoordAxes_Vector[i], coordAxes_CPU_Pinned, coord_Axes_streamBytes, hipMemcpyHostToDevice, stream[i]);
    //     gpuErrchk( hipPeekAtLastError() );

    //     // Run the forward projection kernel
    //     dim3 dimGrid(32, 32, 1);
    //     dim3 dimBlock(4, 4, 1);

    //     gpuForwardProjectKernel<<< dimGrid, dimBlock >>>(
    //         gpuVol_Vector[i], 134, gpuCASImgs_Vector[i],
    //          128, gpuCoordAxes_Vector[i], nAxes,
    //          63, ker_bessel_Vector[i], 501, 2);

    //     // gpuForwardProjectKernel<<< dimGrid, dimBlock >>>(vol, volSize, img, imgSize, axes, nAxes, maskRadius,ker, kerSize, kerHWidth);

    //     gpuErrchk( hipPeekAtLastError() );

    //     // Copy the resulting gpuCASImgs to the host (CPU)
    //     // TO DO: Only copy a subset of this
    //     hipMemcpyAsync(CASImgs_CPU_Pinned, &gpuCASImgs_Vector[i], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost, stream[i]);
    //     gpuErrchk( hipPeekAtLastError() );

    // }

    // gpuErrchk( hipDeviceSynchronize() );

    // std::cout << "Done with gpuForwardProjectKernel" << '\n';

}






   // float *d_test;
    // hipMalloc((void **) &d_test, 3702784*sizeof(float));

    //float *h_test = new float[3702784];

    // float *h_test;
    // h_test = (float *)malloc(3702784*sizeof(float));

        



// // Which memory location should we start the transfer on?
// int streamOffset = i * streamSize;

// int grid_dim = ceil(size/nStreams/blockSize);



// // Run the addition kernel
// AddVectorsMask<<<grid_dim, blockSize, 0, stream[i]>>>(devPtrA, devPtrB, devPtrC, size, streamOffset);

// // Run the square elements kernel
// SquareElements<<<grid_dim, blockSize, 0, stream[i]>>>(devPtrC, size, streamOffset);

// // Run the cosine of elements kernel
// CosineElements<<<grid_dim, blockSize, 0, stream[i]>>>(devPtrC, size, streamOffset);

// // Run the square elements kernel
// SquareElements<<<grid_dim, blockSize, 0, stream[i]>>>(devPtrC, size, streamOffset);

// // Copy the result back to the host
// hipMemcpyAsync(&C[streamOffset], &devPtrC[streamOffset], streamBytes, hipMemcpyDeviceToHost, stream[i]);