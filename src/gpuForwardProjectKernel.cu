#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"
#include <math.h>       /* round, floor, ceil, trunc */
 
__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter= volSize/2;
    int imgCenter=imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;   

    __shared__ float locKer[1000];

       
    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        locKer[kerIndex]=*(ker+kerIndex);
    }
    __syncthreads();      

    // Are we inside the image bounds?
    if ( i < 0 || i > volSize || j < 0 || j > volSize)
    {
        return;
    }

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;

            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        //w=expf(-(ri*ri+rj*rj+rk*rk)/(2*sigma*sigma));  

                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+//w;
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));

                        // }
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i
}

void gpuForwardProject(
    std::vector<float *> gpuVol_Vector, std::vector<float *> gpuCASImgs_Vector,          // Vector of GPU array pointers
    std::vector<float *> gpuCoordAxes_Vector, std::vector<float *> ker_bessel_Vector,    // Vector of GPU array pointers
    float *CASImgs_CPU_Pinned, float *coordAxes_CPU_Pinned,                              // Pointers to pinned CPU arrays for input / output
    int volSize, int imgSize, int nAxes, float maskRadius, int kerSize, float kerHWidth, // kernel Parameters and constants
    int numGPUs, int nStreams, int gridSize, int blockSize,                              // Streaming parameters
	int MaxAxesAllocated
    
)
{ 
    std::cout << "Running gpuForwardProject()..." << '\n';
	
    // Define CUDA kernel dimensions
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    // Create the CUDA streams
	hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t)*nStreams);

    for (int i = 0; i < nStreams; i++) // Loop through the streams
    { 
		int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs
        hipSetDevice(curr_GPU);         
        hipStreamCreate(&stream[i]);
    }

	// Initilize a vector which will remember how many axes have been assigned to each GPU
	// during the current batch. This is needed for calculating the offset for the stream when 
	// the number of streams is greater than the number of GPUs. This resets to zeros after each
	// batch since the same GPU memory is used for the next batch.
	std::vector<int> numAxesGPU_Batch;
	for (int i = 0; i < numGPUs; i++)
	{
		numAxesGPU_Batch.push_back(0);
	}
	
	// How many coordinate axes to assign to each stream?
	int numAxesPerStream;
	if (nAxes <= MaxAxesAllocated)
	{
		// The number of coordinate axes is less than or equal to the total number of axes to process
		numAxesPerStream = ceil((double)nAxes / (double)nStreams);
	}
	else
	{
		// Several batches will be needed so evenly split the MaxAxesAllocated by the number of streams
		numAxesPerStream = ceil((double)MaxAxesAllocated / (double)nStreams);
	}	
                   
    int processed_nAxes = 0; // Cumulative number of axes which have already been assigned to a CUDA stream

	// While we have coordinate axes to process, loop through the GPUs and the streams
	int MaxBatches = 10000; // Maximum iterations in case we get stuck in the while loop
	int batch = 0;

	while (processed_nAxes < nAxes && batch < MaxBatches)
	{
		for (int i = 0; i < nStreams; i++) // Loop through the streams 
		{
            int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs                     
			hipSetDevice(curr_GPU);    // This needs to match the GPU where the stream was created on

            // If we're about to process more than the number of coordinate axes, process the remaining faction of numAxesPerStream
			if (processed_nAxes + numAxesPerStream >= nAxes)
			{
				// Process the remaining fraction of numAxesPerStream
				numAxesPerStream = min(numAxesPerStream, nAxes - processed_nAxes);
			}

			// Check to make sure we don't try to process more coordinate axes than we have and that we have at least one axes to process
			if (processed_nAxes + numAxesPerStream > nAxes || numAxesPerStream < 1)
			{
				return;
			}
						
			// Calculate the offsets (in bytes) to determine which part of the array to copy for this stream
			int CoordAxes_CPU_Offset = processed_nAxes * 9;  // Each axes has 9 elements (X, Y, Z)
			int coord_Axes_CPU_streamBytes = numAxesPerStream * 9 * sizeof(float);

			// Use the number of axes already assigned to this GPU since starting the current batch to calculate the currect offset			
			int gpuCASImgs_Offset = numAxesGPU_Batch[curr_GPU] * imgSize * imgSize;
			int gpuCoordAxes_Stream_Offset = numAxesGPU_Batch[curr_GPU] * 9;

        	// Copy the section of gpuCoordAxes which this stream will process on the current GPU
			hipMemcpyAsync(&gpuCoordAxes_Vector[curr_GPU][gpuCoordAxes_Stream_Offset], &coordAxes_CPU_Pinned[CoordAxes_CPU_Offset], coord_Axes_CPU_streamBytes, hipMemcpyHostToDevice, stream[i]);
                 
            // Run the forward projection kernel     
			gpuForwardProjectKernel <<< dimGrid, dimBlock, 0, stream[i] >> > (
				gpuVol_Vector[curr_GPU], volSize, &gpuCASImgs_Vector[curr_GPU][gpuCASImgs_Offset],
				imgSize, &gpuCoordAxes_Vector[curr_GPU][gpuCoordAxes_Stream_Offset], numAxesPerStream,
				maskRadius, ker_bessel_Vector[curr_GPU], 501, 2);

			// Have to use unsigned long long since the array may be longer than the max value int32 can represent
			// imgSize is the size of the zero padded projection images
			unsigned long long *CASImgs_CPU_Offset = new  unsigned long long[3];
			CASImgs_CPU_Offset[0] = (unsigned long long)(imgSize);
			CASImgs_CPU_Offset[1] = (unsigned long long)(imgSize);
			CASImgs_CPU_Offset[2] = (unsigned long long)(processed_nAxes);

			// How many bytes are the output images?
			int gpuCASImgs_streamBytes = imgSize * imgSize * numAxesPerStream * sizeof(float);
			
			// Lastly, copy the resulting cropped projection images back to the host pinned memory (CPU)
			hipMemcpyAsync(
				&CASImgs_CPU_Pinned[CASImgs_CPU_Offset[0] * CASImgs_CPU_Offset[1] * CASImgs_CPU_Offset[2]],
				&gpuCASImgs_Vector[curr_GPU][gpuCASImgs_Offset], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost, stream[i]);
			
			// Update the overall number of coordinate axes which have already been assigned to a CUDA stream
			processed_nAxes = processed_nAxes + numAxesPerStream;

			// Update the number of axes which have been assigned to this GPU during the current batch
            numAxesGPU_Batch[curr_GPU] = numAxesGPU_Batch[curr_GPU] + numAxesPerStream;
                
        }

		// Reset the number of axes assigned to each gpu to all zeros before starting another batch
		for (int curr_GPU = 0; curr_GPU < numGPUs; curr_GPU++)
		{
			numAxesGPU_Batch[curr_GPU] = 0;
		}

		// Increment the batch number
		batch++;

		// Synchronize before running the next batch
		// TO DO: Consider replacing with hipStreamWaitEvent or similar to prevent blocking of the CPU
		hipDeviceSynchronize();
	}

    // Destroy the streams
    for (int i = 0; i < nStreams; i++) {
        hipStreamDestroy(stream[i]);
    }   

    return; 
}




