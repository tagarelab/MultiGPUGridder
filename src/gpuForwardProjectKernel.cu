#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"
#include <math.h>       /* round, floor, ceil, trunc */
 
__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter= volSize/2;
    int imgCenter=imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;   

    __shared__ float locKer[1000];

       
    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        locKer[kerIndex]=*(ker+kerIndex);
    }
    __syncthreads();      

    // Are we inside the image bounds?
    if ( i < 0 || i > volSize || j < 0 || j > volSize)
    {
        return;
    }

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;

            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        //w=expf(-(ri*ri+rj*rj+rk*rk)/(2*sigma*sigma));  

                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+//w;
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));

                        // }
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i
}

void gpuForwardProject(
    std::vector<float *> gpuVol_Vector, std::vector<float *> gpuCASImgs_Vector,          // Vector of GPU array pointers
    std::vector<float *> gpuCoordAxes_Vector, std::vector<float *> ker_bessel_Vector,    // Vector of GPU array pointers
    // std::vector<hipfftComplex *> gpuComplexImgs_Vector,                                   // Vector of GPU array pointers
    // std::vector<hipfftComplex *> gpuComplexImgs_Shifted_Vector,                           // Vector of GPU array pointers
    float *CASImgs_CPU_Pinned, float *coordAxes_CPU_Pinned,                              // Pointers to pinned CPU arrays for input / output
    int volSize, int imgSize, int nAxes, float maskRadius, int kerSize, float kerHWidth, // kernel Parameters and constants
    int numGPUs, int nStreams, int gridSize, int blockSize, int nBatches,                 // Streaming parameters
    std::vector<int> numAxesPerStream
)
{ 
    std::cout << "Running gpuForwardProject()..." << '\n';

    // Define CUDA kernel dimensions
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    // Create the CUDA streams
	hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t)*nStreams);

    for (int i = 0; i < nStreams; i++) // Loop through the streams
    { 
        int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs
        hipSetDevice(curr_GPU);         
        hipStreamCreate(&stream[i]);
    }

    int processed_nAxes = 0; // Cumulative number of axes which have already been assigned to a CUDA stream

    for (int currBatch = 0; currBatch < nBatches; currBatch++) // Loop through the batches
    {   
        for (int i = 0; i < nStreams; i++) // Loop through the streams   
        {            
            std::cout << "Running stream " << i << " and batch " << currBatch << '\n';

            int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs        
            hipSetDevice(curr_GPU);  
            
            // Check to make sure we don't try to process more coord axes than we have
            if (processed_nAxes + numAxesPerStream[i] >= nAxes) 
            {
                numAxesPerStream[i] = min(numAxesPerStream[i], nAxes - processed_nAxes);
            }

            // Is there at least one coordinate axes to process for this stream?
            if (numAxesPerStream[i] <= 1 ) // TO DO: Fix this && processed_nAxes < nAxes
            {
                continue;
            }
                            
            // Calculate the offsets (in bytes) to determine which part of the array to copy for this stream
            int gpuCoordAxes_Offset    = processed_nAxes * 9 * 1;          // Each axes has 9 elements (X, Y, Z)
            int coord_Axes_streamBytes = numAxesPerStream[i] * 9 * sizeof(float); // Copy the entire vector for now
                
            // Copy the section of gpuCoordAxes which this stream will process on the current GPU
            hipMemcpyAsync(gpuCoordAxes_Vector[i], &coordAxes_CPU_Pinned[gpuCoordAxes_Offset], coord_Axes_streamBytes, hipMemcpyHostToDevice, stream[i]);            

            // Run the forward projection kernel
            // NOTE: Only need one gpuVol_Vector and one ker_bessel_Vector per GPU
            // NOTE: Each stream needs its own gpuCASImgs_Vector and gpuCoordAxes_Vector
            gpuForwardProjectKernel<<< dimGrid, dimBlock, 0, stream[i] >>>(
                gpuVol_Vector[curr_GPU], volSize, gpuCASImgs_Vector[i],
                imgSize, gpuCoordAxes_Vector[i], numAxesPerStream[i],
                maskRadius, ker_bessel_Vector[curr_GPU], 501, 2);    

            // Have to use unsigned long long since the array may be longer than the max value int32 can represent
            // imgSize is the size of the zero padded projection images
            unsigned long long *CASImgs_CPU_Offset = new  unsigned long long[3];
            CASImgs_CPU_Offset[0] = (unsigned long long)(imgSize);
            CASImgs_CPU_Offset[1] = (unsigned long long)(imgSize);
            CASImgs_CPU_Offset[2] = (unsigned long long)(processed_nAxes);
            
            // How many bytes are the output images?
            int gpuCASImgs_streamBytes = imgSize * imgSize * numAxesPerStream[i] * sizeof(float);    

            // Lastly, copy the resulting cropped projection images back to the host pinned memory (CPU)
            hipMemcpyAsync(
                &CASImgs_CPU_Pinned[CASImgs_CPU_Offset[0] * CASImgs_CPU_Offset[1] * CASImgs_CPU_Offset[2]],
                gpuCASImgs_Vector[i], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost, stream[i]);

            // Update the number of coordinate axes which have already been assigned to a CUDA stream
            processed_nAxes = processed_nAxes + numAxesPerStream[i];
            
            std::cout << "processed_nAxes: " << processed_nAxes << '\n';
            std::cout << "Axes remaining: " << nAxes - processed_nAxes << '\n';
        } 
    }

    hipDeviceSynchronize();

    std::cout << "Done with gpuForwardProject()..." << '\n';

    return; 
}













/*                 
            dim3 dimGrid_CAS_to_Imgs(32, 32, nAxes_Stream);
            dim3 dimBlock_CAS_to_Imgs(imgSize/32,imgSize/32,1); 

            std::cout << "gpuImgs_Vector.size(): " << gpuImgs_Vector.size() << '\n';
            std::cout << "stream " << i << '\n';
            
            // hipfftComplex *d_imgs; // TEST
            // hipMalloc(&d_imgs, sizeof(hipfftComplex) * imgSize * imgSize * nAxes_Stream); // TEST

            float * d_CASImgs_test;
            hipMalloc(&d_CASImgs_test, sizeof(float) * imgSize * imgSize * nAxes_Stream);

            // Run the CUDA kernel for transforming the CASImgs to complex imgs (in order to apply the inverse FFT)
            CASImgsToImgs<<< dimGrid_CAS_to_Imgs, dimBlock_CAS_to_Imgs, 0, stream[i] >>>(
                gpuCASImgs_Vector[i], gpuImgs_Vector[i], imgSize
            );
     */

            // Plan the inverse FFT operation (for transforming the CASImgs back to imgs)
            // https://arcb.csc.ncsu.edu/~mueller/cluster/nvidia/0.8/NVIDIA_CUFFT_Library_0.8.pdf
            // https://docs.nvidia.com/cuda/cufft/index.html

       




            // TO DO: Need to apply fftshift before the inverse FFT https://github.com/OrangeOwlSolutions/FFT/wiki/The-fftshift-in-CUDA
            // http://www.orangeowlsolutions.com/archives/251

            // Execute the inverse FFT on each 2D slice of the gpuCASImgs
            //hipfftExecC2C(plan, (hipfftComplex *) gpuImgs_Vector[i], (hipfftComplex *) gpuImgs_Vector[i], HIPFFT_BACKWARD);


            //hipMemcpy(h_imgs, d_imgs, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
            
            //for (int z = 0; z < size; z ++)
            //{
            //    std::cout << "hipfftComplex h_imgs.x[" << z << "]: " << h_imgs[z].x << '\n';
            //    std::cout << "hipfftComplex h_imgs.y[" << z << "]: " << h_imgs[z].y << '\n';
   
           // }









              // Define CUDA kernel dimensions for converting CASImgs to imgs
            // dim3 dimGrid_CAS_to_Imgs(gridSize, gridSize, 1);
            // dim3 dimBlock_CAS_to_Imgs(blockSize, blockSize, nAxes_Stream);
            
            // // Run the CUDA kernel for transforming the CASImgs to complex imgs (in order to apply the inverse FFT)
            // CASImgsToImgs<<< dimGrid_CAS_to_Imgs, dimBlock_CAS_to_Imgs, 0, stream[i] >>>(
            //     gpuCASImgs_Vector[i], gpuImgs_Vector[i], imgSize
            // );
            
                /*
            // Transform the CASImgs to complex float2 type
            int size = 100;
            hipfftComplex *h_complex_array, *h_imgs, *d_imgs;
            float * d_CASImgs_test;
            float * h_CASImgs_test;


            h_CASImgs_test = (float *) malloc(sizeof(float) * size);
            hipMalloc(&d_CASImgs_test, sizeof(float) * size);

            for (int k = 0; k < size; k++) {
                h_CASImgs_test[k] = k;
            }

            hipMalloc(&d_imgs, sizeof(hipfftComplex) * size);


            h_complex_array = (hipfftComplex *) malloc(sizeof(hipfftComplex) * size);
            h_imgs = (hipfftComplex *) malloc(sizeof(hipfftComplex) * size);

            for (int k = 0; k < size; k++) {
                h_complex_array[k].x = k;//rand() / (float) RAND_MAX;
                h_complex_array[k].y = 0;
              }
 
            // Example output array (hipfftReal)
            hipfftComplex *output_test = (hipfftComplex*)malloc(size*sizeof(hipfftComplex));

            */








    // // imgSize = 5;

    // int nRows = 5;
    // int nCols = 5;
    // // int n[2] = {nRows, nCols};
    // // int howMany = 1; //nAxes_Stream

    // int IMAGE_DIM = 5;
    // int NUM_IMGS = 2;

    // int num_real_elements = NUM_IMGS * IMAGE_DIM * IMAGE_DIM; 
    
    // int batch = NUM_IMGS;           // --- Number of batched executions
    // int rank = 2;                   // --- 2D FFTs
    // int n[2] = {nRows, nCols};      // --- Size of the Fourier transform
    // int idist = nRows*nCols;        // --- Distance between batches
    // int odist = nRows*nCols;        // --- Distance between batches

    // int inembed[] = {nRows, nCols}; // --- Input size with pitch
    // int onembed[] = {nRows, nCols}; // --- Output size with pitch

    // int istride = 1;                // --- Distance between two successive input/output elements
    // int ostride = 1;                // --- Distance between two successive input/output elements

    // hipfftPlanMany(&forwardFFTPlan,  rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

    // // ALLOCATE HOST MEMORY
    // float *h_img;
    // hipfftComplex* h_complex_img;
    // h_complex_img = (hipfftComplex*)malloc(num_real_elements * sizeof(hipfftComplex));
    // std::cout << "INPUT" << '\n';
    // for (int x=0; x < IMAGE_DIM; x++)
    // {
    //     for (int y=0; y < IMAGE_DIM; y++)
    //     {
    //         h_complex_img[y*IMAGE_DIM+x].x = x * IMAGE_DIM + y;
    //         std::cout << "h_complex_img[" << x << "][" << y << "].x: " <<  h_complex_img[y*IMAGE_DIM+x].x << '\n';
    //     }
    // }

    // for (int x=0; x < IMAGE_DIM; x++)
    // {
    //     int temp_x = x + IMAGE_DIM*IMAGE_DIM; // offset for image two
    //     for (int y=0; y < IMAGE_DIM; y++)
    //     {
    //         h_complex_img[y*IMAGE_DIM+temp_x].x = x * IMAGE_DIM + y ;
    //         std::cout << "h_complex_img[" << temp_x << "][" << y << "].x: " <<  h_complex_img[y*IMAGE_DIM+temp_x].x << '\n';
    //     }
    // }

    // // DEVICE MEMORY
    // float *d_img;
    // hipfftComplex *d_complex_imgSpec, *d_output;

    // // ALLOCATE DEVICE MEMORY
    // hipMalloc( (void**) &d_complex_imgSpec, num_real_elements * sizeof(hipfftComplex));	
    // hipMalloc( (void**) &d_output, num_real_elements * sizeof(hipfftComplex));

    // // copy host memory to device (input image)
    // hipMemcpy( d_complex_imgSpec, h_complex_img, num_real_elements * sizeof(hipfftComplex), hipMemcpyHostToDevice);        

    // // now run the forward FFT on the device (real to complex)
    // hipfftExecC2C(forwardFFTPlan, (hipfftComplex *) d_complex_imgSpec, (hipfftComplex *) d_output, HIPFFT_FORWARD);

    // hipDeviceSynchronize();
    // // hipfftExecR2C(forwardFFTPlan, d_img, d_complex_imgSpec, HIPFFT_FORWARD);

    // // copy the DEVICE complex data to the HOST
    // // NOTE: we are only doing this so that you can see the data -- in general you want
    // // to do your computation on the GPU without wasting the time of copying it back to the host
    // hipMemcpy( h_complex_img, d_output, num_real_elements * sizeof(hipfftComplex), hipMemcpyDeviceToHost) ;
    // hipDeviceSynchronize();
    
    // std::cout << "" << '\n';
    // std::cout << "" << '\n';
    // std::cout << "" << '\n';
    // std::cout << "OUTPUT" << '\n';
    // std::cout << "IMAGE ONE" << '\n';
    // for (int x=0; x < (IMAGE_DIM); x++)
    // {
    //     std::cout << "h_complex_img[" << x << "]: ";
    //     for (int y=0; y < IMAGE_DIM; y++)
    //     {
    //         if ((h_complex_img[y*IMAGE_DIM+x].x*h_complex_img[y*IMAGE_DIM+x].x) < 0.001)
    //         {
    //             std::cout << " "   <<  0  << " + " << h_complex_img[y*IMAGE_DIM+x].y << "i   ";
    //         } else
    //         {
    //             std::cout << " "   <<  h_complex_img[y*IMAGE_DIM+x].x  << " + " << h_complex_img[y*IMAGE_DIM+x].y << "i   ";
    //         }
          
    //     }
    //     std::cout << '\n';
    // }

    // std::cout << '\n';
    // std::cout << '\n';
    // std::cout << '\n';
    // std::cout << "IMAGE TWO" << '\n';
    // for (int x=0; x < (IMAGE_DIM); x++)
    // {
    //     // Offset is IMAGE_DIM * IMAGE_DIM since we are on image two now
    //     int temp_x = x + IMAGE_DIM * IMAGE_DIM;
        
    //     std::cout << "h_complex_img[" << x << "]: ";
    //     for (int y=0; y < IMAGE_DIM; y++)
    //     {     
    //         //std::cout << "y*IMAGE_DIM+temp_x: " << y*IMAGE_DIM+temp_x << '\n';

            
    //         if ((h_complex_img[y*IMAGE_DIM+temp_x].x*h_complex_img[y*IMAGE_DIM+temp_x].x) < 0.001)
    //         {
    //             std::cout << " "   <<  0  << " + " << h_complex_img[y*IMAGE_DIM+temp_x].y << "i   ";
    //         } else
    //         {
    //             std::cout << " "   <<  h_complex_img[y*IMAGE_DIM+temp_x].x  << " + " << h_complex_img[y*IMAGE_DIM+temp_x].y << "i   ";
    //         }
          
    //     }
    //     std::cout << '\n';
    // }

