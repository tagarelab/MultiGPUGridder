#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"
#include <math.h>       /* round, floor, ceil, trunc */
 
__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter= volSize/2;
    int imgCenter=imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;   

    __shared__ float locKer[1000];

       
    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        locKer[kerIndex]=*(ker+kerIndex);
    }
    __syncthreads();      

    // Are we inside the image bounds?
    if ( i < 0 || i > volSize || j < 0 || j > volSize)
    {
        return;
    }

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;

            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        //w=expf(-(ri*ri+rj*rj+rk*rk)/(2*sigma*sigma));  

                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+//w;
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));

                        // }
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i
}

void gpuForwardProject(gpuGridder * gridder)
{ 
    std::cout << "Running gpuForwardProject()..." << '\n';
	
    // Copy the relevent parameters to a local variable to make the code cleaner
    int gridSize = gridder->GetGridSize();
    int blockSize = gridder->GetBlockSize();
    int nAxes = gridder->GetNumAxes();
    int MaxAxesAllocated = gridder->GetMaxAxesAllocated();
    int nStreams = gridder->GetNumStreams();
    int GPU_Device = gridder->GetGPUDevice();
    int* imgSizePtr = gridder->GetImgSize();    
    int* volSizePtr = gridder->GetVolumeSize();
    float maskRadius = gridder->GetMaskRadius();


    int imgSize  = imgSizePtr[0]; // The volume must be a square for now so just use the first dimension
    int volSize = volSizePtr[0];  // The volume must be a square for now so just use the first dimension

    // Pointers to memory already allocated on the GPU (i.e. the device)
    float * d_CASVolume = gridder->GetCASVolumePtr();
    float * d_CASImgs = gridder->GetCASImgsPtr();
    float * d_CoordAxes = gridder->GetCoordAxesPtr();
    float * d_KB_Table = gridder->GetKBTablePtr();

    // Pointers to pinned CPU memory
    float * coordAxes_CPU_Pinned = gridder->GetCoordAxesPtr();
    float * CASImgs_CPU_Pinned = gridder->GetCASImgsPtr_CPU();

    // CUDA streams
    hipStream_t *streams = gridder->GetStreamsPtr();
    
    // Set the current GPU device to run the kernel
    hipSetDevice(GPU_Device);    

    // Define CUDA kernel dimensions
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

	// How many coordinate axes to assign to each stream?
	int numAxesPerStream;
	if (nAxes <= MaxAxesAllocated)
	{
		// The number of coordinate axes is less than or equal to the total number of axes to process
		numAxesPerStream = ceil((double)nAxes / (double)nStreams);
	}
	else
	{
		// Several batches will be needed so evenly split the MaxAxesAllocated by the number of streams
		numAxesPerStream = ceil((double)MaxAxesAllocated / (double)nStreams);
	}	
                   
    int processed_nAxes = 0; // Cumulative number of axes which have already been assigned to a CUDA stream

	// While we have coordinate axes to process, loop through the GPUs and the streams
	int MaxBatches = 1000; // Maximum iterations in case we get stuck in the while loop for some reason
	int batch = 0;

	while (processed_nAxes < nAxes && batch < MaxBatches)
	{
		for (int i = 0; i < nStreams; i++) // Loop through the streams 
		{
            // If we're about to process more than the number of coordinate axes, process the remaining faction of numAxesPerStream
			if (processed_nAxes + numAxesPerStream >= nAxes)
			{
				// Process the remaining fraction of numAxesPerStream
				numAxesPerStream = min(numAxesPerStream, nAxes - processed_nAxes);
			}

			// Check to make sure we don't try to process more coordinate axes than we have and that we have at least one axes to process
			if (processed_nAxes + numAxesPerStream > nAxes || numAxesPerStream < 1)
			{
				return;
			}
						
			// Calculate the offsets (in bytes) to determine which part of the array to copy for this stream
			int CoordAxes_CPU_Offset = processed_nAxes * 9;  // Each axes has 9 elements (X, Y, Z)
			int coord_Axes_CPU_streamBytes = numAxesPerStream * 9 * sizeof(float);

			// Use the number of axes already assigned to this GPU since starting the current batch to calculate the currect offset			
			int gpuCASImgs_Offset = processed_nAxes * imgSize * imgSize;
			int gpuCoordAxes_Stream_Offset = processed_nAxes * 9;

        	// Copy the section of gpuCoordAxes which this stream will process on the current GPU
            hipMemcpyAsync(
                &d_CoordAxes[gpuCoordAxes_Stream_Offset], 
                &coordAxes_CPU_Pinned[CoordAxes_CPU_Offset],
                coord_Axes_CPU_streamBytes,
                hipMemcpyHostToDevice, streams[i]);
                 
            // Run the forward projection kernel     
			gpuForwardProjectKernel <<< dimGrid, dimBlock, 0, streams[i] >> > (
				d_CASVolume, volSize, &d_CASImgs[gpuCASImgs_Offset],
				imgSize, &d_CoordAxes[gpuCoordAxes_Stream_Offset], numAxesPerStream,
				maskRadius, d_KB_Table, 501, 2);

			// Have to use unsigned long long since the array may be longer than the max value int32 can represent
			// imgSize is the size of the zero padded projection images
			unsigned long long *CASImgs_CPU_Offset = new  unsigned long long[3];
			CASImgs_CPU_Offset[0] = (unsigned long long)(imgSize);
			CASImgs_CPU_Offset[1] = (unsigned long long)(imgSize);
			CASImgs_CPU_Offset[2] = (unsigned long long)(processed_nAxes);

			// How many bytes are the output images?
			int gpuCASImgs_streamBytes = imgSize * imgSize * numAxesPerStream * sizeof(float);
			
			// Lastly, copy the resulting cropped projection images back to the host pinned memory (CPU)
			hipMemcpyAsync(
				&CASImgs_CPU_Pinned[CASImgs_CPU_Offset[0] * CASImgs_CPU_Offset[1] * CASImgs_CPU_Offset[2]],
				&d_CASImgs[gpuCASImgs_Offset], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost, streams[i]);
			
			// Update the overall number of coordinate axes which have already been assigned to a CUDA stream
			processed_nAxes = processed_nAxes + numAxesPerStream;
                
        }

		// Increment the batch number
		batch++;

		// Synchronize before running the next batch
		// TO DO: Consider replacing with hipStreamWaitEvent or similar to prevent blocking of the CPU
		// hipDeviceSynchronize();
	}


    return; 
}




