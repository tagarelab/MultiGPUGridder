#include "hip/hip_runtime.h"
#include "gpuFFT.h"

// Constructor
gpuFFT::gpuFFT(/* args */)
{
}

// Deconstructor
gpuFFT::~gpuFFT()
{
}

template <typename T>
__global__ void cufftShift_3D_slice_kernel(T* data, int N, int nSlices)
{
    // In place FFT shift using GPU
    // Modified from https://raw.githubusercontent.com/marwan-abdellah/cufftShift/master/Src/CUDA/Kernels/in-place/cufftShift_3D_IP.cu
    // GNU Lesser General Public License

    // 3D Volume & 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;
    int sVolume = N * N * N;

    // Transformations Equations
    int sEq1 = (sVolume + sSlice + sLine) / 2;
    int sEq2 = (sVolume + sSlice - sLine) / 2;
    int sEq3 = (sVolume - sSlice + sLine) / 2;
    int sEq4 = (sVolume - sSlice - sLine) / 2;

    // Thread
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth  = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index 2D
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Are we within the image bounds?
    if (xIndex < 0 || xIndex >= N || yIndex < 0 || yIndex >= N )
    {
        return;
    }

    T regTemp;

	// Each thread will do all the slices for some X, Y position in the 3D matrix
	for (int zIndex = 0; zIndex < nSlices; zIndex++)
	{

        // Thread Index Converted into 1D Index
        int index = (zIndex * sSlice) + (yIndex * sLine) + xIndex;

        if (zIndex < N / 2)
        {
            if (xIndex < N / 2)
            {
                if (yIndex < N / 2)
                {
                    regTemp = data[index];

                    // First Quad
                    data[index] = data[index + sEq1];

                    // Fourth Quad
                    data[index + sEq1] = regTemp;
                }
                else
                {
                    regTemp = data[index];

                    // Third Quad
                    data[index] = data[index + sEq3];

                    // Second Quad
                    data[index + sEq3] = regTemp;
                }
            }
            else
            {
                if (yIndex < N / 2)
                {
                    regTemp = data[index];

                    // Second Quad
                    data[index] = data[index + sEq2];

                    // Third Quad
                    data[index + sEq2] = regTemp;
                }
                else
                {
                    regTemp = data[index];

                    // Fourth Quad
                    data[index] = data[index + sEq4];

                    // First Quad
                    data[index + sEq4] = regTemp;
                }
            }
        }
    }
}

__global__ void ComplexImgsToCASImgs(float* CASimgs, hipfftComplex* imgs, int imgSize)
{
	// CUDA kernel for converting CASImgs to imgs
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
	int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

	// Are we outside the bounds of the image?
	if (i >= imgSize || i < 0 || j >= imgSize || j < 0) {
		return;
	}

	// Each thread will do all the slices for position X and Y
	for (int k = 0; k < imgSize; k++)
	{
		// CASimgs is the same dimensions as imgs
		int ndx = i + j * imgSize + k * imgSize * imgSize;

		// Summation of the real and imaginary components
		CASimgs[ndx] = imgs[ndx].x + imgs[ndx].y;
	}

	return;
}

__global__ void CASImgsToComplexImgs(float* CASimgs, hipfftComplex* imgs, int imgSize, int nSlices)
{
    // CUDA kernel for converting CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the image?
    if (i >= imgSize || i < 0 || j >= imgSize || j < 0){
        return;
    }
    
    // Each thread will do all the slices for position X and Y
    for (int k=0; k < nSlices; k++)  {        

        // CASimgs is the same dimensions as imgs
        int ndx_1 = i + j * imgSize + k * imgSize * imgSize;
        
        // Skip the first row and first column
        if (i == 0 || j == 0)
        {
            // Real component
            imgs[ndx_1].x = 0;

            // Imaginary component
            imgs[ndx_1].y = 0;

            
        } else 
        {

            // Offset to skip the first row then subtract from the end of the matrix and add the offset where the particular image starts in CASimgs
            int ndx_2 = imgSize + imgSize * imgSize - (i + j * imgSize) + k * imgSize * imgSize;

            // Real component
            imgs[ndx_1].x = 0.5*(CASimgs[ndx_1] + CASimgs[ndx_2]);

            // Imaginary component
            imgs[ndx_1].y = 0.5*(CASimgs[ndx_1] - CASimgs[ndx_2]);

        }
    }

    return;
}

__global__ void ComplexToReal(hipfftComplex* ComplexImg, float* RealImg, int imgSize, int nSlices)
{
    // CUDA kernel for extracting the real component of a hipfftComplex and then save the real component to a float array

    // CUDA kernel for converting the CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row

    // Are we outside the bounds of the image?
    if (i >= imgSize || i < 0 || j >= imgSize || j < 0){
        return;
    }
    
    // Each thread will do all the slices for some position X and Y
    for (int k=0; k < nSlices; k++)
    {         
        // Get the linear index of the current position
        int ndx = i + j * imgSize + k * imgSize * imgSize;       

        RealImg[ndx] = ComplexImg[ndx].x;
    }
}

__global__ void PadVolumeKernel(float* input, float* output, int inputImgSize, int outputImgSize, int padding)
{
    // Zero pad a volume using the GPU

    // Index of the output (smaller) image
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row
    
    // Are we outside the bounds of the image?
    if (i >= inputImgSize || i < 0 || j >= inputImgSize || j < 0){
        return;
    }

    // // Iterate over the input image (i.e. the smaller image)
    for (int k = 0; k < inputImgSize; k++){  

        // Get the linear index of the output (smaller) image
        int ndx_1 = i + j * inputImgSize + k * inputImgSize * inputImgSize;   

        // Get the linear index of the output (larger) image    
        int ndx_2 = 
        (i + padding) + 
        (j + padding) * outputImgSize +
        (k + padding) * outputImgSize *  outputImgSize;  

        output[ndx_2] = input[ndx_1];
    }
}

float *gpuFFT::PadVolume(float *inputVol, int inputImgSize, int outputImgSize)
{
    // Pad a volume (of dimensions 3) with zeros
    // Note: Output volume is larger than the input volume

    // Check the input parameters
    if(inputImgSize <=0)
    {
        std::cerr << "CropVolume(): Invalid image size." << '\n';
    }

    // Create the output volume
    float *outputVol = new float[outputImgSize * outputImgSize * outputImgSize];
    memset(outputVol, 0, outputImgSize * outputImgSize * outputImgSize * sizeof(float));

    // for (int i = 0; i < outputImgSize * outputImgSize * outputImgSize; i++)
    // {
    //     outputVol[i] = 0;
    // }

    // How much to crop on each side?
    int padding = (outputImgSize - inputImgSize) / 2;

    // For very small matrix sizes it might be faster to use the CPU instead of the GPU
    bool use_gpu = true;

    if (use_gpu == true)
    {
        // Allocate GPU memory to hold the input and output arrays
        float *d_input; 
        hipMalloc(&d_input, sizeof(float) * inputImgSize * inputImgSize * inputImgSize);
        float *d_output; 
        hipMalloc(&d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize);

        // Copy the input volume to the device
        hipMemcpy(d_input, inputVol, sizeof(float) * inputImgSize * inputImgSize * inputImgSize, hipMemcpyHostToDevice);

        // Run kernel to pad the intput array
        int gridSize = 32;
        int blockSize = ceil(inputImgSize / gridSize);

        dim3 dimGridCrop(gridSize, gridSize, 1);
        dim3 dimBlockCrop(blockSize, blockSize, 1);

        PadVolumeKernel<<< dimGridCrop, dimBlockCrop >>>(d_input, d_output, inputImgSize, outputImgSize, padding);

        // Copy the result back to the host
        hipMemcpy(outputVol, d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize, hipMemcpyDeviceToHost);

        // Free the GPU memory
        hipFree(d_input);
        hipFree(d_output);

    } else 
    {
        // Iterate over the input image (i.e. the smaller image)
        for (int i = 0; i < inputImgSize; i++)
        {
            for (int j = 0; j < inputImgSize; j++)
            {
                for (int k = 0; k < inputImgSize; k++)
                {

                    int input_ndx = i + j*inputImgSize + k*inputImgSize*inputImgSize;

                    int output_ndx = (i + padding) + (j+padding)*outputImgSize + (k+padding)*outputImgSize*outputImgSize;

                    outputVol[output_ndx] = inputVol[input_ndx];
                }
            }
        }
    }
    
    return outputVol;

}

float* gpuFFT::VolumeToCAS(float* inputVol, int inputVolSize, int interpFactor, int extraPadding)
{
    // Convert a CUDA array to CAS array
    // Note: The volume must be square (i.e. have the same dimensions for the X, Y, and Z)
    // Step 1: Pad with zeros
    // Step 2: fftshift
    // Step 3: Take discrete Fourier transform using cuFFT
    // Step 4: fftshift
    // Step 5: Convert to CAS volume using CUDA kernel
    
    // STEP 1
    // Example: input size = 128; interpFactor = 2; extra padding = 3; -> padded size = 262
    int paddedVolSize = inputVolSize * interpFactor;

    // Pad the input volume with zeros
    float* inputVol_Padded = PadVolume(inputVol, inputVolSize, paddedVolSize);
    
    // Plan the forward FFT
    hipfftHandle forwardFFTPlan;           
    hipfftPlan3d(&forwardFFTPlan, paddedVolSize, paddedVolSize, paddedVolSize, HIPFFT_C2C);

    int array_size = paddedVolSize * paddedVolSize * paddedVolSize;
    
    // Allocate memory for the resulting CAS volumes
    float * d_CAS_Vol, *h_CAS_Vol;
    hipMalloc(&d_CAS_Vol, sizeof(float) * array_size);
    h_CAS_Vol = (float *) malloc(sizeof(float) * array_size);

    // Create temporary arrays to hold the hipfftComplex array        
    hipfftComplex *h_complex_array, *d_complex_array;
    hipMalloc(&d_complex_array, sizeof(hipfftComplex) * array_size);
    h_complex_array = (hipfftComplex *) malloc(sizeof(hipfftComplex) * array_size);
    
    // Convert the padded volume to a hipfftComplex array
    // TO DO: Replace this with the CUDA kernel
    for (int k = 0; k < array_size; k++) {
        h_complex_array[k].x = inputVol_Padded[k]; // Real component
        h_complex_array[k].y = 0;                  // Imaginary component
    }

    // Copy the complex version of the GPU volume to the first GPU
    hipMemcpy( d_complex_array, h_complex_array, array_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);        

    int gridSize  = ceil(paddedVolSize / 32);
    int blockSize = 32; // i.e. 32*32 threads

    // Define CUDA kernel dimensions for converting the complex volume to a CAS volume
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    // STEP 2
    // Apply an in place 3D FFT Shift
    cufftShift_3D_slice_kernel<<< dimGrid, dimBlock >>> (d_complex_array, paddedVolSize, paddedVolSize);
   
    // STEP 3
    // Execute the forward FFT on the 3D array
    hipfftExecC2C(forwardFFTPlan, (hipfftComplex *) d_complex_array, (hipfftComplex *) d_complex_array, HIPFFT_FORWARD);

    // STEP 4
    // Apply a second in place 3D FFT Shift
    cufftShift_3D_slice_kernel<<< dimGrid, dimBlock >>> (d_complex_array, paddedVolSize, paddedVolSize);

    // STEP 5
    // Convert the complex result of the forward FFT to a CAS img type
    ComplexImgsToCASImgs<<< dimGrid, dimBlock >>>(
        d_CAS_Vol, d_complex_array, paddedVolSize
    );
    
    // Copy the resulting CAS volume back to the host
    hipMemcpy(h_CAS_Vol, d_CAS_Vol, array_size * sizeof(float), hipMemcpyDeviceToHost);        

    // STEP 6
    // Pad the result with the additional padding
    int paddedVolSize_Extra = paddedVolSize + extraPadding * 2;

    // Pad the padded volume with the extra zero padding
    float *outputVol = PadVolume(h_CAS_Vol, paddedVolSize, paddedVolSize_Extra);

    // Free the temporary memory
    hipFree(d_complex_array);
    hipFree(d_CAS_Vol);   
    std::free(inputVol_Padded);
    std::free(h_CAS_Vol);

    // Return the resulting CAS volume
    return outputVol;

}