#include "hip/hip_runtime.h"
#include "gpuFFT.h"

gpuFFT::gpuFFT(/* args */)
{
}

gpuFFT::~gpuFFT()
{
}


__global__ void PadVolumeKernel(float* input, float* output, int intputImgSize, int outputImgSize, int padding)
{
    // Zero pad a volume using the GPU

    // Index of the output (smaller) image
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row
    

    // Are we outside the bounds of the image?
    if (i >= intputImgSize || i < 0 || j >= intputImgSize || j < 0){
        return;
    }


    // // Iterate over the input image (i.e. the smaller image)
    for (int k = 0; k < intputImgSize; k++){

        // Get the linear index of the output (smaller) image
        int ndx_1 = i + j * intputImgSize + k * intputImgSize * intputImgSize;   

        // Get the linear index of the output (larger) image    
        int ndx_2 = 
        (i + padding) + 
        (j + padding) * outputImgSize +
        (k + padding) * outputImgSize *  outputImgSize;  

        output[ndx_1] = input[ndx_2];
    }
}


float *gpuFFT::PadVolume(float *inputVol, int inputImgSize, int outputImgSize)
{
    // Pad a volume (of dimensions 3) with zeros
    // Note: Output volume is larger than the input volume

    // Check the input parameters
    if(inputImgSize <=0)
    {
        std::cerr << "CropVolume(): Invalid image size." << '\n';
    }

    // Create the output volume
    //int outputImgSize = interpFactor * inputImgSize;

    float *outputVol = new float[outputImgSize * outputImgSize * outputImgSize];

    for (int i = 0; i < outputImgSize * outputImgSize * outputImgSize; i++)
    {
        outputVol[i] = 0; // Initilize the output volume to zeros first
    }

    std::cout << "Output volume size: " << outputImgSize << '\n';

    // How much to crop on each side?
    int padding = (outputImgSize - inputImgSize) / 2;

    bool use_gpu = true;

    if (use_gpu == true)
    {
        std::cout << "Padding: " << padding << '\n';
        std::cout << "inputImgSize: " << inputImgSize << '\n';
        std::cout << "outputImgSize: " << outputImgSize << '\n';

        // Allocate GPU memory to hold the input and output arrays
        float *d_input; 
        hipMalloc(&d_input, sizeof(float) * inputImgSize * inputImgSize * inputImgSize);
        float *d_output; 
        hipMalloc(&d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize);


        // Run kernel to pad the intput array
        int gridSize = 32;
        int blockSize = inputImgSize / gridSize;

        dim3 dimGridCrop(gridSize, gridSize, 1);
        dim3 dimBlockCrop(blockSize, blockSize, 1);

        PadVolumeKernel<<< dimGridCrop, dimBlockCrop>>>(d_input, d_output, inputImgSize, outputImgSize, padding);

        // Copy the result back to the host
        hipMemcpy(outputVol, d_output, sizeof(float) * outputImgSize * outputImgSize * outputImgSize, hipMemcpyDeviceToHost);

        // Free the GPU memory
        hipFree(d_input);
        hipFree(d_output);

    } else 
    {
        // Iterate over the input image (i.e. the smaller image)
        for (int i = 0; i < inputImgSize; i++)
        {
            for (int j = 0; j < inputImgSize; j++)
            {
                for (int k = 0; k < inputImgSize; k++)
                {

                    int input_ndx = i + j*inputImgSize + k*inputImgSize*inputImgSize;

                    int output_ndx = (i + padding) + (j+padding)*outputImgSize + (k+padding)*outputImgSize*outputImgSize;

                    outputVol[output_ndx] = inputVol[input_ndx];
                }
            }
        }
    }
    
    return outputVol;

}