#include "hip/hip_runtime.h"
#include "gpuForwardProject.h"

__global__ void gpuForwardProjectKernel(const float* vol, int volSize, float* img,int imgSize, float *axes, int nAxes,float maskRadius,
    float* ker, int kerSize, float kerHWidth)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int volCenter= volSize/2;
    int imgCenter=imgSize/2;
    float f_vol_i,f_vol_j,f_vol_k;
    int img_i;
    float *img_ptr;
    int int_vol_i,int_vol_j,int_vol_k;
    int i1,j1,k1;//,kerIndex;
    float r=sqrtf( (float) (i-imgCenter)*(i-imgCenter)+(j-imgCenter)*(j-imgCenter));
    float *nx,*ny;
    int convW=roundf(kerHWidth);
    float ri,rj,rk,w;
    //float sigma=0.33*convW;
    float wi,wj,wk;
    float kerCenter=((float)kerSize-1)/2;
    float kerScale=kerCenter/kerHWidth;
    int kerIndex;   

    __shared__ float locKer[1000];

    if (threadIdx.x==0)
    {
        /* Copy over the kernel */
        for (kerIndex=0;kerIndex<kerSize;kerIndex++) 
        locKer[kerIndex]=*(ker+kerIndex);
    }
    __syncthreads();      

    for(img_i=0;img_i<nAxes;img_i++)
    {
        img_ptr=img+img_i*imgSize*imgSize;

        if (r<=maskRadius)
        {
            nx=axes+9*img_i;
            ny=nx+3;

            f_vol_i= (*(nx))*((float)(i-imgCenter))+(*(ny))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_j= (*(nx+1))*((float)(i-imgCenter))+(*(ny+1))*((float)(j-imgCenter))+(float)volCenter;
            f_vol_k= (*(nx+2))*((float)(i-imgCenter))+(*(ny+2))*((float)(j-imgCenter))+(float)volCenter;

            int_vol_i= roundf(f_vol_i);
            int_vol_j= roundf(f_vol_j);
            int_vol_k= roundf(f_vol_k);

            *(img_ptr+j*imgSize+i)=0;
            
            for (i1=int_vol_i-convW;i1<=int_vol_i+convW;i1++)
            {
                ri= (float)i1-f_vol_i;
                ri=min(max(ri,(float)-convW),(float)convW);
                kerIndex=roundf( ri*kerScale+kerCenter);
                kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wi=*(ker+kerIndex);
                wi=*(locKer+kerIndex);

                for (j1=int_vol_j-convW;j1<=int_vol_j+convW;j1++)
                {

                    rj= (float)j1-f_vol_j;
                    rj=min(max(rj,(float)-convW),(float)convW);
                    kerIndex=roundf( rj*kerScale+kerCenter);
                    kerIndex=min(max(kerIndex,0),kerSize-1);
                //  wj=*(ker+kerIndex);
                    wj=*(locKer+kerIndex);

                    for (k1=int_vol_k-convW;k1<=int_vol_k+convW;k1++)
                    {
                        rk= (float)k1-f_vol_k;
                        rk=min(max(rk,(float)-convW),(float)convW);
                        kerIndex=roundf( rk*kerScale+kerCenter);
                        kerIndex=min(max(kerIndex,0),kerSize-1);
                    //   wk=*(ker+kerIndex);
                        wk=*(locKer+kerIndex);
                        w=wi*wj*wk;

                        //w=expf(-(ri*ri+rj*rj+rk*rk)/(2*sigma*sigma));
                        *(img_ptr+j*imgSize+i)=*(img_ptr+j*imgSize+i)+//w;
                                w*( *(vol+k1*volSize*volSize+j1*volSize+i1));
                    } //End k1
                }//End j1   
            }//End i1
        }//End if r
    }//End img_i
}

__global__ void CASImgsToImgs(float* CASimgs, hipfftComplex* imgs, int imgSize)
{
    // CUDA kernel for converting the CASImgs to imgs
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Row
    int k = blockIdx.z * blockDim.z + threadIdx.z; // Which image?

    // For now, CASimgs is the same dimensions as imgs
    int ndx_1 = i + j * imgSize + k * imgSize * imgSize;
    
    // Skip the first row and first column
    if (i == 0 || j == 0)
    {
        // Real component
        imgs[ndx_1].x = 0;

        // Imaginary component
        imgs[ndx_1].y = 0;

        return;
    }

    // Are we outside the bounds of the image?
    if (i >= imgSize || i < 0 || j >= imgSize || j < 0){
        return;
    }

    // Offset to skip the first row then subtract from the end of the matrix and add the offset where the particular image starts in CASimgs
    int ndx_2 = imgSize + imgSize * imgSize - ndx_1 + k * imgSize * imgSize;

    // Real component
    imgs[ndx_1].x = 0.5*(CASimgs[ndx_1] + CASimgs[ndx_2]);

    // Imaginary component
    imgs[ndx_1].y = 0.5*(CASimgs[ndx_1] - CASimgs[ndx_2]);

    return;
}




void gpuForwardProject(
    std::vector<float*> gpuVol_Vector, std::vector<float*> gpuCASImgs_Vector,       // Vector of GPU array pointers
    std::vector<float*> gpuCoordAxes_Vector, std::vector<float*> ker_bessel_Vector, // Vector of GPU array pointers
    float * CASImgs_CPU_Pinned, float * coordAxes_CPU_Pinned, // Pointers to pinned CPU arrays for input / output
    int volSize, int imgSize, int nAxes, float maskRadius, int kerSize, float kerHWidth, // kernel Parameters and constants
    int numGPUs, int nStreams, int gridSize, int blockSize, int nBatches // Streaming parameters
)
{
    std::cout << "Running gpuForwardProject()..." << '\n';
    
    // Define CUDA kernel dimensions
    dim3 dimGrid(gridSize, gridSize, 1);
    dim3 dimBlock(blockSize, blockSize, 1);

    // Create the CUDA streams
	hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t)*nStreams);
    //hipStream_t stream[nStreams]; 		

    for (int i = 0; i < nStreams; i++) // Loop through the streams
    { 
        int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs
        hipSetDevice(curr_GPU);         
        hipStreamCreate(&stream[i]);
    }

    int processed_nAxes = 0; // Cumulative number of axes which have already been assigned to a CUDA stream

    for (int currBatch = 0; currBatch < nBatches; currBatch++) // Loop through the batches
    {   
        for (int i = 0; i < nStreams; i++) // Loop through the streams 
        {             
            int curr_GPU = i % numGPUs; // Use the remainder operator to split evenly between GPUs
        
            hipSetDevice(curr_GPU);     

            // How many coordinate axes to assign to this CUDA stream? 
            int nAxes_Stream = ceil((double)nAxes / (nBatches * nStreams)); // Ceil needed if nStreams is not a multiple of numGPUs            

            // Check to make sure we don't try to process more coord axes than we have
            if (processed_nAxes + nAxes_Stream > nAxes) 
            {
                // Process the remaining streams (at least one axes is left)
                nAxes_Stream = nAxes_Stream - (processed_nAxes + nAxes_Stream - nAxes); // Remove the extra axes that are past the maximum nAxes
            }
            
            // Is there at least one coordinate axes to process for this stream?
            if (nAxes_Stream < 1)
            {
                continue; // Otherwise skip this stream
            }  
                    
            // Calculate the offsets (in bytes) to determine which part of the array to copy for this stream
            int gpuCoordAxes_Offset    = processed_nAxes * 9 * 1;          // Each axes has 9 elements (X, Y, Z)
            int coord_Axes_streamBytes = nAxes_Stream * 9 * sizeof(float); // Copy the entire vector for now

            // Use unsigned long long int type to allow for array length larger than maximum int32 value 
            // Number of bytes of already processed images
            // Have to use unsigned long long since the array may be longer than the max value int32 can represent
            unsigned long long *CASImgs_CPU_Offset = new  unsigned long long[3];
            CASImgs_CPU_Offset[0] = (unsigned long long)(imgSize);
            CASImgs_CPU_Offset[1] = (unsigned long long)(imgSize);
            CASImgs_CPU_Offset[2] = (unsigned long long)(processed_nAxes);
            
            // How many bytes are the output images?
            int gpuCASImgs_streamBytes = imgSize * imgSize * nAxes_Stream * sizeof(float);          
            
            // Copy the section of gpuCoordAxes which this stream will process on the current GPU
            hipMemcpyAsync(gpuCoordAxes_Vector[i], &coordAxes_CPU_Pinned[gpuCoordAxes_Offset], coord_Axes_streamBytes, hipMemcpyHostToDevice, stream[i]);
            
              // Define CUDA kernel dimensions for converting CASImgs to imgs
            // dim3 dimGrid_CAS_to_Imgs(gridSize, gridSize, 1);
            // dim3 dimBlock_CAS_to_Imgs(blockSize, blockSize, nAxes_Stream);
            
            // // Run the CUDA kernel for transforming the CASImgs to complex imgs (in order to apply the inverse FFT)
            // CASImgsToImgs<<< dimGrid_CAS_to_Imgs, dimBlock_CAS_to_Imgs, 0, stream[i] >>>(
            //     gpuCASImgs_Vector[i], gpuImgs_Vector[i], imgSize
            // );
            
                /*
            // Transform the CASImgs to complex float2 type
            int size = 100;
            hipfftComplex *h_complex_array, *h_imgs, *d_imgs;
            float * d_CASImgs_test;
            float * h_CASImgs_test;


            h_CASImgs_test = (float *) malloc(sizeof(float) * size);
            hipMalloc(&d_CASImgs_test, sizeof(float) * size);

            for (int k = 0; k < size; k++) {
                h_CASImgs_test[k] = k;
            }

            hipMalloc(&d_imgs, sizeof(hipfftComplex) * size);


            h_complex_array = (hipfftComplex *) malloc(sizeof(hipfftComplex) * size);
            h_imgs = (hipfftComplex *) malloc(sizeof(hipfftComplex) * size);

            for (int k = 0; k < size; k++) {
                h_complex_array[k].x = k;//rand() / (float) RAND_MAX;
                h_complex_array[k].y = 0;
              }
 
            // Example output array (hipfftReal)
            hipfftComplex *output_test = (hipfftComplex*)malloc(size*sizeof(hipfftComplex));


            
            // Run the forward projection kernel
            // NOTE: Only need one gpuVol_Vector and one ker_bessel_Vector per GPU
            // NOTE: Each stream needs its own gpuCASImgs_Vector and gpuCoordAxes_Vector
            gpuForwardProjectKernel<<< dimGrid, dimBlock, 0, stream[i] >>>(
                gpuVol_Vector[curr_GPU], volSize, gpuCASImgs_Vector[i],
                imgSize, gpuCoordAxes_Vector[i], nAxes_Stream,
                maskRadius, ker_bessel_Vector[curr_GPU], 501, 2);        

            // Copy the resulting gpuCASImgs to the host (CPU)
            hipMemcpyAsync(
                &CASImgs_CPU_Pinned[CASImgs_CPU_Offset[0] * CASImgs_CPU_Offset[1] * CASImgs_CPU_Offset[2]],
                gpuCASImgs_Vector[i], gpuCASImgs_streamBytes, hipMemcpyDeviceToHost, stream[i]);

            // Update the number of coordinate axes which have already been assigned to a CUDA stream
            processed_nAxes = processed_nAxes + nAxes_Stream;
        } 

        // hipDeviceSynchronize();
    }

    hipDeviceSynchronize();

    std::cout << "Done with gpuForwardProject()..." << '\n';

    return; 
}
